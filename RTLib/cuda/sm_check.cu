#include <hip/hip_runtime.h>
#include <iostream>
int main(){
    hipDeviceProp_t deviceProp = {};
    hipGetDeviceProperties(&deviceProp,0);
    std::cout << "arch=" << deviceProp.major << deviceProp.minor << "\n";
    return 0;
}