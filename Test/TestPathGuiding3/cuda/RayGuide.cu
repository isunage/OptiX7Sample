#include "hip/hip_runtime.h"
#define __HIPCC__
#include "RayTrace.h"
#include "PathGuiding.h"
struct RadiancePRD {
    DTreeWrapper* dTree;
    float3        dTreeVoxelSize;
    float3        radiance;
    float3        bsdfVal;
    float3        throughPut;
    float         woPdf, bsdfPdf, dTreePdf;
    float         cosine;
    float         distance;
    unsigned int  seed;
    bool          isDelta;
    bool          countEmitted;
    bool          done;
};
extern "C" {
    __constant__ RayTraceParams params;
}
static __forceinline__ __device__ float3       faceForward(const float3& n, const float3& i, const float3& nref) {
    return copysignf(1.0f, rtlib::dot(n, i)) * nref;
}
static __forceinline__ __device__ void*        unpackPointer(unsigned int p0, unsigned int p1) {
    return reinterpret_cast<void*>(rtlib::to_combine(p0, p1));
}
static __forceinline__ __device__ void         packPointer(void* ptr, unsigned int& p0, unsigned int& p1) {
    const unsigned long long llv = reinterpret_cast<const unsigned long long>(ptr);
    p0 = rtlib::to_upper(llv);
    p1 = rtlib::to_lower(llv);
}
static __forceinline__ __device__ float3       unpackFloat3(unsigned int p0, unsigned p1, unsigned int p2)
{
    return make_float3(__uint_as_float(p0), __uint_as_float(p1), __uint_as_float(p2));
}
static __forceinline__ __device__ void         packFloat3(const float3& v, unsigned int& p0, unsigned& p1, unsigned int& p2)
{
    p0 = __float_as_uint(v.x);
    p1 = __float_as_uint(v.y);
    p2 = __float_as_uint(v.z);
}
static __forceinline__ __device__ RadiancePRD* getRadiancePRD() {
    unsigned int p0 = optixGetPayload_0();
    unsigned int p1 = optixGetPayload_1();
    return static_cast<RadiancePRD*>(unpackPointer(p0, p1));
}
static __forceinline__ __device__ void         setRadiancePRD(RadiancePRD* prd) {
    unsigned int p0;
    unsigned int p1;
    packPointer(static_cast<void*>(prd), p0, p1);
    optixSetPayload_0(p0);
    optixSetPayload_1(p1);
}
static __forceinline__ __device__ void         setRayOrigin(const float3& origin) {
    unsigned int p2, p3, p4;
    packFloat3(origin, p2, p3, p4);
    optixSetPayload_2(p2);
    optixSetPayload_3(p3);
    optixSetPayload_4(p4);
}
static __forceinline__ __device__ void         setRayDirection(const float3& direction) {
    unsigned int p5, p6, p7;
    packFloat3(direction, p5, p6, p7);
    optixSetPayload_5(p5);
    optixSetPayload_6(p6);
    optixSetPayload_7(p7);
}
static __forceinline__ __device__ void         setPayloadOccluded(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}
static __forceinline__ __device__ void         traceRadiance(
    OptixTraversableHandle handle,
    float3& rayOrigin,
    float3& rayDirection,
    float tmin, float tmax,
    RadiancePRD* prd) {
    unsigned int p0, p1, p2, p3, p4, p5, p6, p7;
    packPointer(prd, p0, p1);
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, RAY_TYPE_RADIANCE, RAY_TYPE_COUNT, RAY_TYPE_RADIANCE, p0, p1, p2, p3, p4, p5, p6, p7);
    rayOrigin = unpackFloat3(p2, p3, p4);
    rayDirection = unpackFloat3(p5, p6, p7);
}
static __forceinline__ __device__ bool         traceOccluded(
    OptixTraversableHandle handle,
    const float3& rayOrigin,
    const float3& rayDirection,
    float tmin, float tmax) {
    unsigned int occluded = false;
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, RAY_TYPE_OCCLUSION, RAY_TYPE_COUNT, RAY_TYPE_OCCLUSION, occluded);
    return occluded;
}
extern "C" __global__ void __raygen__def() {

    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    auto* rgData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u = rgData->u;
    const float3 v = rgData->v;
    const float3 w = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.radiance = make_float3(0.0f);
        prd.bsdfVal  = make_float3(1.0f);
        prd.throughPut = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.cosine       = 0.0f;
        prd.distance     = 0.0f;
        prd.done         = false;
        prd.countEmitted = true;
        prd.isDelta      = false;
        prd.seed         = seed;
        int depth = 0;
        for (;;) {
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            //vertices�̍X�V
            //Radiance�̍X�V
            //Result�̍X�V
            result += prd.radiance;
            //ThroughPut�̍X�V
            bool isValidThroughPut = (!isnan(prd.throughPut.x) && !isnan(prd.throughPut.y) && !isnan(prd.throughPut.z) &&
                isfinite(prd.throughPut.x) && isfinite(prd.throughPut.y) && isfinite(prd.throughPut.z) &&
                (prd.throughPut.x >= 0.0f && prd.throughPut.y >= 0.0f && prd.throughPut.z >= 0.0f));
            bool isValidDirection  = (!isnan(rayDirection.x) && !isnan(rayDirection.y) && !isnan(rayDirection.z));
            if (prd.done || depth >= params.maxTraceDepth || !isValidThroughPut || !isValidDirection) {
                break;
            }
            depth++;
        }
        seed = prd.seed;
    } while(--i);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor = prevAccumColor + result;
        float3 frameColor = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
        params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __raygen__pg_def() {

    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    auto* rgData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u = rgData->u;
    const float3 v = rgData->v;
    const float3 w = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    TraceVertex vertices[RAY_TRACE_MAX_VERTEX_COUNT] = {};
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.radiance    = make_float3(0.0f);
        prd.bsdfVal     = make_float3(1.0f);
        prd.throughPut  = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.distance     = 0.0f;
        prd.done         = false;
        prd.isDelta      = false;
        prd.countEmitted = true;
        prd.seed = seed;
        int depth = 0;
        for (;;) {
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            if(!params.isFinal){
                vertices[depth].rayOrigin      = rayOrigin;
                vertices[depth].rayDirection   = rayDirection;
                vertices[depth].dTree          = prd.dTree;
                vertices[depth].dTreeVoxelSize = prd.dTreeVoxelSize;
                vertices[depth].throughPut     = prd.throughPut;
                vertices[depth].bsdfVal        = prd.bsdfVal;
                vertices[depth].radiance       = make_float3(0.0f);
                vertices[depth].woPdf          = prd.woPdf;
                vertices[depth].bsdfPdf        = prd.bsdfPdf;
                vertices[depth].dTreePdf       = prd.dTreePdf;
                vertices[depth].cosine         = prd.cosine;
                vertices[depth].isDelta        = prd.isDelta;
                for (int j = 0; j < depth; ++j) {
                    vertices[j].Record(prd.radiance);
                }
            }
            //OK
            //Result�̍X�V
            result += prd.radiance;
            bool isValidThroughPut = (!isnan(prd.throughPut.x) && !isnan(prd.throughPut.y) && !isnan(prd.throughPut.z) &&
                                    isfinite(prd.throughPut.x) && isfinite(prd.throughPut.y) && isfinite(prd.throughPut.z) &&
                                            (prd.throughPut.x >= 0.0f && prd.throughPut.y >= 0.0f && prd.throughPut.z >= 0.0f));
            bool isValidDirection  = (!isnan(rayDirection.x) && !isnan(rayDirection.y) && !isnan(rayDirection.z));
            if (prd.done || depth >= params.maxTraceDepth || !isValidThroughPut || !isValidDirection) {
                break;
            }
            depth++;
        }
        if(!params.isFinal){
            for (int j = 0; j < depth; ++j) {
                vertices[j].Commit<RAY_TRACE_S_FILTER, RAY_TRACE_D_FILTER>(params.sdTree, 1.0f);
            }
        }
        seed = prd.seed;
    } while(--i);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor     = prevAccumColor + result;
        float3 frameColor           = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor                  = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
        params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __raygen__pg_nee() {
    const uint3 idx   = optixGetLaunchIndex();
    const uint3 dim   = optixGetLaunchDimensions();
    auto* rgData      = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u    = rgData->u;
    const float3 v    = rgData->v;
    const float3 w    = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    TraceVertex vertices[RAY_TRACE_MAX_VERTEX_COUNT] = {};
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.radiance = make_float3(0.0f);
        prd.bsdfVal = make_float3(1.0f);
        prd.throughPut = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.distance = 0.0f;
        prd.done = false;
        prd.isDelta = false;
        prd.countEmitted = true;
        prd.seed = seed;
        int depth = 0;
        for (;;) {
            if (depth >= params.maxTraceDepth) {
                prd.done = true;
            }
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            if (!params.isFinal) {
                vertices[depth].rayOrigin    = rayOrigin;
                vertices[depth].rayDirection = rayDirection;
                vertices[depth].dTree = prd.dTree;
                vertices[depth].dTreeVoxelSize = prd.dTreeVoxelSize;
                vertices[depth].throughPut = prd.throughPut;
                vertices[depth].bsdfVal = prd.bsdfVal;
                vertices[depth].radiance = make_float3(0.0f);
                vertices[depth].woPdf = prd.woPdf;
                vertices[depth].bsdfPdf = prd.bsdfPdf;
                vertices[depth].dTreePdf = prd.dTreePdf;
                vertices[depth].cosine = prd.cosine;
                vertices[depth].isDelta = prd.isDelta;
                for (int j = 0; j < depth; ++j) {
                    vertices[j].Record(prd.radiance);
                }
            }
            //OK
            //Result�̍X�V
            result += prd.radiance;

            bool isValidThroughPut = (!isnan(prd.throughPut.x) && !isnan(prd.throughPut.y)   && !isnan(prd.throughPut.z)   &&
                                    isfinite(prd.throughPut.x) && isfinite(prd.throughPut.y) && isfinite(prd.throughPut.z) &&
                                     (prd.throughPut.x >= 0.0f &&   prd.throughPut.y >= 0.0f && prd.throughPut.z >= 0.0f));
            bool isValidDirection  = (!isnan(rayDirection.x)   && !isnan(rayDirection.y)     && !isnan(rayDirection.z));

            if (prd.done || !isValidThroughPut || !isValidDirection) {
                break;
            }
            depth++;
        }
        if (!params.isFinal) {
            for (int j = 0; j < depth; ++j) {
                vertices[j].Commit<RAY_TRACE_S_FILTER, RAY_TRACE_D_FILTER>(params.sdTree, 1.0f);
            }
        }
        seed = prd.seed;
    } while (--i);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor = prevAccumColor + result;
        float3 frameColor = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
        params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __miss__radiance() {
    auto* msData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    RadiancePRD* prd = getRadiancePRD();
    prd->radiance = make_float3(msData->bgColor.x, msData->bgColor.y, msData->bgColor.z) * prd->throughPut * static_cast<float>(prd->countEmitted);
    prd->dTree    = nullptr;
    prd->woPdf    = 1.0f;
    prd->bsdfPdf  = 0.0f;
    prd->dTreePdf = 0.0f;
    prd->bsdfVal  = make_float3(1.0f);
    prd->cosine   = 0.0f;
    prd->distance = optixGetRayTmax();
    prd->done     = true;
}
extern "C" __global__ void __miss__occluded() {
    setPayloadOccluded(false);
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();

    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;

    const auto diffuse = hgData->getDiffuseColor(texCoord);
    const auto emission = hgData->getEmissionColor(texCoord);

    const auto distance   = optixGetRayTmax();
    const float3 position = optixGetWorldRayOrigin() + distance * rayDirection;
    float3 newDirection   = make_float3(0.0f);
    RadiancePRD* prd      = getRadiancePRD();

    prd->dTree    = nullptr;
    prd->radiance = emission * prd->throughPut;
    prd->distance = distance;

    rtlib::Xorshift32 xor32(prd->seed);
    rtlib::ONB onb(normal);
    newDirection = onb.local(rtlib::random_cosine_direction(xor32));

    const auto cosine = rtlib::dot(newDirection, normal);

    prd->bsdfPdf  = fabsf(cosine) / RTLIB_M_PI;
    prd->dTreePdf = 0.0f;
    prd->woPdf    = prd->bsdfPdf;

    setRayOrigin(position);
    setRayDirection(newDirection);

    prd->cosine      = cosine;
    prd->bsdfVal     = diffuse / RTLIB_M_PI;
    prd->throughPut *= diffuse;
    prd->seed        = xor32.m_seed;

    prd->countEmitted= true;
    prd->isDelta     = false;
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_nee() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();

    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;

    const auto diffuse = hgData->getDiffuseColor(texCoord);
    const auto emission = hgData->getEmissionColor(texCoord);

    const auto distance = optixGetRayTmax();
    const auto position = optixGetWorldRayOrigin() + distance * rayDirection;
    float3 newDirection = make_float3(0.0f);
    RadiancePRD* prd = getRadiancePRD();

    const auto prvThroughPut = prd->throughPut;
    prd->dTree = nullptr;
    prd->radiance = emission * prvThroughPut * static_cast<float>(prd->countEmitted);
    prd->distance = distance;
    if (prd->done) {
        return;
    }
    rtlib::Xorshift32 xor32(prd->seed);
    rtlib::ONB onb(normal);
    newDirection = onb.local(rtlib::random_cosine_direction(xor32));

    const auto cosine = rtlib::dot(newDirection, normal);

    prd->bsdfPdf = fabsf(cosine) / RTLIB_M_PI;
    prd->dTreePdf = 0.0f;
    prd->woPdf = prd->bsdfPdf;

    setRayOrigin(position);
    setRayDirection(newDirection);

    prd->cosine = cosine;
    prd->bsdfVal = diffuse / RTLIB_M_PI;
    prd->throughPut *= diffuse;

    prd->countEmitted = false;
    prd->isDelta = false;

    {
        const float2 z = rtlib::random_float2(xor32);
        const auto   light = params.light;
        const float3 lightPos = light.corner + light.v1 * z.x + light.v2 * z.y;
        const float  Ldist = rtlib::distance(lightPos, position);
        const float3 lightDir = rtlib::normalize(lightPos - position);
        const float  ndl = rtlib::dot(normal, lightDir);
        const float  lndl = -rtlib::dot(light.normal, lightDir);
        float weight = 0.0f;
        if (ndl > 0.0f && lndl > 0.0f) {
            const bool occluded = traceOccluded(params.gasHandle, position, lightDir, 0.01f, Ldist - 0.01f);
            if (!occluded) {
                //printf("not Occluded!\n");
                const float A = rtlib::length(rtlib::cross(light.v1, light.v2));
                weight = ndl * lndl * A / (Ldist * Ldist);
            }
        }
        prd->radiance += light.emission * prvThroughPut * weight * diffuse / RTLIB_M_PI;
    }

    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_pg_def() {

    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();

    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;

    const auto diffuse = hgData->getDiffuseColor(texCoord);
    const auto emission = hgData->getEmissionColor(texCoord);

    const auto distance  = optixGetRayTmax();
    const auto position  = optixGetWorldRayOrigin() + distance * rayDirection;
    auto  dTreeVoxelSize = make_float3(0.0f);
    const auto dTree     = params.sdTree.GetDTreeWrapper(position, dTreeVoxelSize);

    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float  cosine1      = 0.0f;
    float  cosine2      = 0.0f;

    RadiancePRD* prd    = getRadiancePRD();
    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->radiance       = emission * prd->throughPut;
    prd->distance       = distance;
    prd->bsdfPdf        = 0.0f;
    prd->bsdfVal        = diffuse / RTLIB_M_PI;
    prd->isDelta        = false;
    rtlib::Xorshift32 xor32(prd->seed);

    setRayOrigin(position);
    if(params.isBuilt){
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(normal, newDirection1);

        if (isnan(newDirection1.x) || isnan(newDirection1.y) || isnan(newDirection1.z)) {
            printf("newDirection1 is nan: new Direction1 = (%f, %f, %f) normal = (%f, %f, %f) n0 = (%f, %f, %f)\n", newDirection1.x, newDirection1.y, newDirection1.z, normal.x, normal.y, normal.z, n0.x, n0.y, n0.z);
        }
    }
    {
        rtlib::ONB onb(normal);
        newDirection2 = onb.local(rtlib::random_cosine_direction(xor32));
        cosine2 = rtlib::dot(normal, newDirection2);
        if (isnan(newDirection2.x) || isnan(newDirection2.y) || isnan(newDirection2.z))
        {
            printf("newDirection2 is nan!\n");
        }
    }

    const float rnd          = rtlib::random_float1(xor32);
    const auto  newDirection = rnd < 0.5f ? newDirection1 : newDirection2;
    const auto  cosine       = rnd < 0.5f ? cosine1 : cosine2;
    const auto  bsdfPdf      = rtlib::max(cosine / RTLIB_M_PI, 0.0f);
    //両方とも正なら
    if (params.isBuilt) {
        const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
        const auto  woPdf = 0.5f * bsdfPdf + 0.5f * dTreePdf;
        prd->bsdfPdf     = bsdfPdf;
        prd->dTreePdf    = dTreePdf;
        prd->woPdf       = woPdf;
        prd->throughPut *= (prd->bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
        setRayDirection(newDirection);
        prd->cosine = cosine;
    }
    else {
        prd->bsdfPdf     = fabsf(cosine2) / RTLIB_M_PI;
        prd->dTreePdf    = 0.0f;
        prd->woPdf       = prd->bsdfPdf;
        prd->throughPut *= (diffuse);
        setRayDirection(newDirection2);
        prd->cosine      = cosine2;
    }
    prd->countEmitted    = true;
    prd->seed            = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_pg_nee() {

    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();

    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;

    const auto diffuse = hgData->getDiffuseColor(texCoord);
    const auto emission = hgData->getEmissionColor(texCoord);

    const auto distance = optixGetRayTmax();
    const auto position = optixGetWorldRayOrigin() + distance * rayDirection;
    auto  dTreeVoxelSize = make_float3(0.0f);
    const auto dTree = params.sdTree.GetDTreeWrapper(position, dTreeVoxelSize);

    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float  cosine1 = 0.0f;
    float  cosine2 = 0.0f;

    RadiancePRD* prd    = getRadiancePRD();
    auto prvThroughPut  = prd->throughPut;
    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->radiance       = emission * prvThroughPut*static_cast<float>(prd->countEmitted);
    prd->distance       = distance;
    prd->bsdfPdf        = 0.0f;
    prd->bsdfVal        = diffuse / RTLIB_M_PI;
    prd->isDelta        = false;
    //new
    if (prd->done) {
        return;
    }
    rtlib::Xorshift32 xor32(prd->seed);
    setRayOrigin(position);
    if (params.isBuilt) {
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(normal, newDirection1);

        if (isnan(newDirection1.x) || isnan(newDirection1.y) || isnan(newDirection1.z)) {
            printf("newDirection1 is nan: new Direction1 = (%f, %f, %f) normal = (%f, %f, %f) n0 = (%f, %f, %f)\n", newDirection1.x, newDirection1.y, newDirection1.z, normal.x, normal.y, normal.z, n0.x, n0.y, n0.z);
        }
    }
    {
        rtlib::ONB onb(normal);
        newDirection2 = onb.local(rtlib::random_cosine_direction(xor32));
        cosine2 = rtlib::dot(normal, newDirection2);
        if (isnan(newDirection2.x) || isnan(newDirection2.y) || isnan(newDirection2.z))
        {
            printf("newDirection2 is nan!\n");
        }
    }

    const float rnd = rtlib::random_float1(xor32);
    const auto  newDirection = rnd < 0.5f ? newDirection1 : newDirection2;
    const auto  cosine = rnd < 0.5f ? cosine1 : cosine2;
    const auto  bsdfPdf = rtlib::max(cosine / RTLIB_M_PI, 0.0f);
    //両方とも正なら
    if (params.isBuilt) {
        const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
        const auto  woPdf    = 0.5f * bsdfPdf + 0.5f * dTreePdf;
        prd->bsdfPdf = bsdfPdf;
        prd->dTreePdf = dTreePdf;
        prd->woPdf = woPdf;
        prd->throughPut *= (prd->bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
        setRayDirection(newDirection);
        prd->cosine = cosine;
    }
    else {
        prd->bsdfPdf = fabsf(cosine2) / RTLIB_M_PI;
        prd->dTreePdf = 0.0f;
        prd->woPdf = prd->bsdfPdf;
        prd->throughPut *= (diffuse);
        setRayDirection(newDirection2);
        prd->cosine = cosine2;
    }
    {
        const float2 z        = rtlib::random_float2(xor32);
        const auto   light    = params.light;
        const float3 lightPos = light.corner + light.v1 * z.x + light.v2 * z.y;
        const float  Ldist    = rtlib::distance(lightPos, position);
        const float3 lightDir = rtlib::normalize(lightPos - position);
        const float ndl       = rtlib::dot(normal, lightDir);
        const float lndl      =-rtlib::dot(light.normal, lightDir);
        const float A         = rtlib::length(rtlib::cross(light.v1, light.v2));
        float lightPdf        = 0.0f;
        float weight          = 0.0f;
        if (ndl > 0.0f && lndl > 0.0f && A>0.0f) {
            const bool occluded = traceOccluded(params.gasHandle, position, lightDir, 0.01f, Ldist - 0.01f);
            if (!occluded) {
                //printf("not Occluded!\n");
                lightPdf      = (Ldist * Ldist) / (lndl * A);
                weight        = ndl * lndl * A / (Ldist * Ldist);
            }
        }
        prd->radiance += light.emission * prvThroughPut * weight * diffuse/RTLIB_M_PI;
    }
    prd->countEmitted = false;
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_specular() {

    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float2 barycentric = optixGetTriangleBarycentrics();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    float3       n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    if (hgData->normals) {
        const float3 nv0 = hgData->normals[hgData->indices[primitiveID].x];
        const float3 nv1 = hgData->normals[hgData->indices[primitiveID].y];
        const float3 nv2 = hgData->normals[hgData->indices[primitiveID].z];
        const bool isValidNv0 = !((nv0.x == 0.0f) && (nv0.y == 0.0f) && (nv0.z == 0.0f));
        const bool isValidNv1 = !((nv1.x == 0.0f) && (nv1.y == 0.0f) && (nv1.z == 0.0f));
        const bool isValidNv2 = !((nv2.x == 0.0f) && (nv2.y == 0.0f) && (nv2.z == 0.0f));
        if (isValidNv0 && isValidNv1 && isValidNv2)
        {
            float3 nv = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize((1.0f - barycentric.x - barycentric.y) * nv0 + barycentric.x * nv1 + barycentric.y * nv2));
            if (rtlib::dot(nv, n0) > 0.0f) {
                n0 = nv;
            }
        }
    }
    const auto normal = n0;
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd = getRadiancePRD();
    prd->dTree    = nullptr;
    prd->radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd->distance = optixGetRayTmax();
    {
        float3 specular = hgData->getSpecularColor(texCoord);
        float3 reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);
        auto cosine = rtlib::dot(reflectDir, normal);

        prd->woPdf = 0.0f;
        prd->dTreePdf = 0.0f;
        prd->bsdfPdf = std::fabsf(cosine);

        setRayOrigin(position);
        setRayDirection(reflectDir);
        prd->cosine = cosine;

        prd->bsdfVal = specular;
        prd->throughPut *= prd->bsdfVal;
        prd->countEmitted = true;
        prd->isDelta      = true;
    }
}
extern "C" __global__ void __closesthit__radiance_for_refraction() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float2 barycentric = optixGetTriangleBarycentrics();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    float3       n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    if (hgData->normals) {
        const float3 nv0 = hgData->normals[hgData->indices[primitiveID].x];
        const float3 nv1 = hgData->normals[hgData->indices[primitiveID].y];
        const float3 nv2 = hgData->normals[hgData->indices[primitiveID].z];
        const bool isValidNv0 = !((nv0.x == 0.0f) && (nv0.y == 0.0f) && (nv0.z == 0.0f));
        const bool isValidNv1 = !((nv1.x == 0.0f) && (nv1.y == 0.0f) && (nv1.z == 0.0f));
        const bool isValidNv2 = !((nv2.x == 0.0f) && (nv2.y == 0.0f) && (nv2.z == 0.0f));
        if (isValidNv0 && isValidNv1 && isValidNv2)
        {
            float3 nv = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize((1.0f - barycentric.x - barycentric.y) * nv0 + barycentric.x * nv1 + barycentric.y * nv2));
            if (rtlib::dot(nv, n0) > 0.0f) {
                n0 = nv;
            }
        }
    }
    float3 normal = {};
    float  refInd = 0.0f;
    if (rtlib::dot(n0, rayDirection) < 0.0f) {
        normal = n0;
        refInd = 1.0f / hgData->refrInd;
    }
    else {
        normal = make_float3(-n0.x, -n0.y, -n0.z);
        refInd = hgData->refrInd;
    }
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd = getRadiancePRD();
    prd->dTree    = nullptr;
    prd->radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd->distance = optixGetRayTmax();
    rtlib::Xorshift32 xor32(prd->seed);
    float3 diffuse = hgData->getDiffuseColor(texCoord);
    float3 specular = hgData->getSpecularColor(texCoord);
    float3 transmit = hgData->transmit;
    {
        float3 reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);
        float  cosine_i = -rtlib::dot(normal, rayDirection);
        float  sine_o_2 = (1.0f - rtlib::pow2(cosine_i)) * rtlib::pow2(refInd);
        float  f0 = rtlib::pow2((1 - refInd) / (1 + refInd));
        float  fresnell = f0 + (1.0f - f0) * rtlib::pow5(1.0f - cosine_i);

        if (rtlib::random_float1(0.0f, 1.0f, xor32) < fresnell || sine_o_2 > 1.0f) {
            float cosine = rtlib::dot(reflectDir, normal);
            prd->woPdf = prd->dTreePdf = 0.0f;
            prd->bsdfPdf = std::fabsf(cosine);
            //printf("reflect: %lf %lf %lf\n", reflectDir.x, reflectDir.y, reflectDir.z);
            setRayOrigin(position + 0.001f * normal);
            setRayDirection(reflectDir);
            prd->cosine = cosine;
            prd->bsdfVal = specular;
            prd->throughPut *= prd->bsdfVal;
        }
        else {
            float  cosine_o = sqrtf(1.0f - sine_o_2);
            float3 k = (rayDirection + cosine_i * normal) / sqrtf(1.0f - cosine_i * cosine_i);
            float3 refractDir = rtlib::normalize(sqrtf(sine_o_2) * k - cosine_o * normal);
            float cosine = rtlib::dot(refractDir, normal);
            prd->woPdf = prd->dTreePdf = 0.0f;
            prd->bsdfPdf = std::fabsf(cosine);
            //printf("refract: %lf %lf %lf\n", refractDir.x, refractDir.y, refractDir.z);
            setRayOrigin(position - 0.001f * normal);
            setRayDirection(refractDir);
            prd->cosine = cosine;
            prd->bsdfVal = make_float3(1.0f);
            prd->throughPut *= prd->bsdfVal;
        }
        prd->isDelta = true;
    }
    prd->countEmitted = true;
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_emission() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    const float3 n0 = rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0));
   // const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);
    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const auto distance = optixGetRayTmax();
    const float3 position = optixGetWorldRayOrigin() + distance * rayDirection;
    RadiancePRD* prd = getRadiancePRD();
    prd->radiance = hgData->getEmissionColor(texCoord) * prd->throughPut * static_cast<float>(prd->countEmitted)*static_cast<float>(rtlib::dot(n0, rayDirection) < 0.0f);
    prd->bsdfVal = make_float3(1.0f);
    prd->woPdf = 0.0f;
    prd->bsdfPdf = 0.0f;
    prd->dTreePdf = 0.0f;
    prd->dTree = nullptr;
    prd->cosine = 0.0f;
    prd->distance = distance;
    prd->done = true;
}
extern "C" __global__ void __closesthit__occluded() {
    setPayloadOccluded(true);
}
extern "C" __global__ void __closesthit__radiance_for_phong_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();

    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];

    const auto reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);

    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const auto diffuse = hgData->getDiffuseColor(texCoord);
    const auto specular = hgData->getSpecularColor(texCoord);
    const auto shinness = hgData->shinness;
    const auto emission = hgData->getEmissionColor(texCoord);
    const auto distance = optixGetRayTmax();
    const float3 position = optixGetWorldRayOrigin() + distance * rayDirection;
    RadiancePRD* prd = getRadiancePRD();

    prd->dTree    = nullptr;
    prd->radiance = emission * prd->throughPut;
    prd->distance = distance;

    rtlib::Xorshift32 xor32(prd->seed);

    auto  newDirection = make_float3(0.0f);
    auto  cosine = 0.0f;

    const auto rnd = rtlib::random_float1(xor32);
    const auto a_diffuse = (diffuse.x + diffuse.y + diffuse.z) / 3.0f;
    const auto a_specular = (specular.x + specular.y + specular.z) / 3.0f;

    if (rnd < a_diffuse) {
        rtlib::ONB onb(normal);
        newDirection = onb.local(rtlib::random_cosine_direction(xor32));
        cosine = rtlib::dot(newDirection, normal);
        prd->bsdfVal = diffuse / (a_diffuse * RTLIB_M_PI);
        prd->bsdfPdf = fabsf(cosine) / RTLIB_M_PI;
        prd->dTreePdf = 0.0f;
        prd->woPdf = prd->bsdfPdf;
        prd->throughPut *= (diffuse / a_diffuse);
        prd->cosine = cosine;
    }
    else if (rnd < a_diffuse + a_specular) {
        const auto cosTht = powf(rtlib::random_float1(0.0f, 1.0f, xor32), 1.0f / (shinness + 1.0f));
        const auto sinTht = sqrtf(1.0f - cosTht * cosTht);
        const auto phi    = rtlib::random_float1(0.0f, RTLIB_M_2PI, xor32);
        rtlib::ONB onb(reflectDir);
        newDirection     = onb.local(make_float3(sinTht * cosf(phi), sinTht * sinf(phi), cosTht));
        cosine           = rtlib::dot(newDirection, normal);
        prd->bsdfVal     = (specular / a_specular) * (shinness + 2.0f) * powf(rtlib::max(rtlib::dot(reflectDir, newDirection),0.0f), shinness) / RTLIB_M_2PI;
        prd->bsdfPdf     = (shinness + 2.0f) * powf(rtlib::max(rtlib::dot(reflectDir, newDirection), 0.0f), shinness) / RTLIB_M_2PI;
        prd->dTreePdf    = 0.0f;
        prd->woPdf       = prd->bsdfPdf;
        prd->throughPut *= (specular * rtlib::max(cosine, 0.0f) / a_specular);
        prd->cosine      = cosine;
    }
    else {
        //printf("Hit!\n");
        //反射しない
        prd->bsdfVal = make_float3(1.0f);
        prd->woPdf = 0.0f;
        prd->bsdfPdf = 0.0f;
        prd->dTreePdf = 0.0f;
        prd->cosine = 0.0f;
        prd->throughPut = make_float3(0.0f);
        prd->dTree = nullptr;
        prd->done  = true;
    }

    setRayOrigin(position);
    setRayDirection(newDirection);
    prd->seed         = xor32.m_seed;
    prd->countEmitted = true;
    prd->isDelta      = false;
}
extern "C" __global__ void __closesthit__radiance_for_phong_pg_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const auto rayDirection = optixGetWorldRayDirection();
    const auto primitiveID = optixGetPrimitiveIndex();

    const auto v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const auto v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const auto v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const auto n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const auto normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const auto barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];


    const auto reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);

    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const auto emission = hgData->getEmissionColor(texCoord);
    const auto diffuse  = hgData->getDiffuseColor(texCoord);
    const auto specular = hgData->getSpecularColor(texCoord);
    const auto shinness = hgData->shinness;
    const auto distance = optixGetRayTmax();
    const auto position = optixGetWorldRayOrigin() + distance * rayDirection;
    //direction
    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float3 newDirection3 = make_float3(0.0f);
    //cosine
    float  cosine1 = 0.0f;
    float  cosine2 = 0.0f;
    float  cosine3 = 0.0f;
    //payLoad
    RadiancePRD* prd    = getRadiancePRD();
    auto dTreeVoxelSize = make_float3(0.0f);
    const auto dTree    = params.sdTree.GetDTreeWrapper(position, dTreeVoxelSize);
    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->radiance       = emission*prd->throughPut;
    prd->distance       = distance;
    prd->isDelta        = false;
    rtlib::Xorshift32 xor32(prd->seed);
    //const auto isValid = false;
    setRayOrigin(position);
    if(params.isBuilt){
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(normal, newDirection1);
    }
    {
        rtlib::ONB onb(normal);
        newDirection2 = onb.local(rtlib::random_cosine_direction(xor32));
        cosine2 = rtlib::dot(normal, newDirection2);

    }
    {
        rtlib::ONB onb(reflectDir);
        const auto cosTht = powf(rtlib::random_float1(0.0f, 1.0f, xor32), 1.0f / (shinness + 1.0f));
        const auto sinTht = sqrtf(1.0f - cosTht * cosTht);
        const auto phi = rtlib::random_float1(0.0f, RTLIB_M_2PI, xor32);
        newDirection3 = onb.local(make_float3(sinTht * cosf(phi), sinTht * sinf(phi), cosTht));
        cosine3 = rtlib::dot(normal, newDirection3);
    }
    const auto  a_diffuse  = (diffuse.x + diffuse.y + diffuse.z) / 3.0f;
    const auto  a_specular = (specular.x + specular.y + specular.z) / 3.0f;
    const float rnd1       = rtlib::random_float1(xor32);
    const float rnd2       = rtlib::random_float1(xor32);

    if (rnd1 < a_diffuse) {
        const auto  newDirection = rnd2 < 0.5f ? newDirection1 : newDirection2;
        const auto  cosine       = rnd2 < 0.5f ?       cosine1 :       cosine2;
        const auto  bsdfPdf      = rtlib::max(cosine / RTLIB_M_PI     , 0.0f);
        const auto  bsdfVal      = diffuse / (RTLIB_M_PI * a_diffuse);
        //両方とも正なら
        if (params.isBuilt) {
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf = 0.5f * dTreePdf + 0.5f * bsdfPdf;
            prd->bsdfVal  = bsdfVal;
            prd->dTreePdf = dTreePdf;
            prd->bsdfPdf  = bsdfPdf;
            prd->woPdf    = woPdf;
            prd->throughPut *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine = cosine;
            setRayDirection(newDirection);
        }
        else {
            prd->bsdfVal = (diffuse / (RTLIB_M_PI * a_diffuse));
            prd->bsdfPdf = fabsf(cosine2) / RTLIB_M_PI;
            prd->dTreePdf = 0.0f;
            prd->woPdf = prd->bsdfPdf;
            prd->throughPut *= (diffuse / a_diffuse);
            prd->cosine = cosine2;
            setRayDirection(newDirection2);
        }
    }
    else if (rnd1 < a_diffuse + a_specular)
    {
        const auto  newDirection = rnd2 < 0.5f ? newDirection1 : newDirection3;
        const auto  cosine       = rnd2 < 0.5f ? cosine1       : cosine3;
        const auto  bsdfPdf      = (shinness + 2.0f) * powf(rtlib::max(rtlib::dot(reflectDir, newDirection), 0.0f), shinness) / RTLIB_M_2PI;
        const auto  bsdfVal      = specular * bsdfPdf / a_specular;
        //両方とも正なら
        if (params.isBuilt)
        {
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf    = 0.5f * dTreePdf + 0.5f * bsdfPdf;
            //printf("Hit1! %f %f\n", woPdf,dTreePdf);
            prd->dTreePdf        = dTreePdf;
            prd->bsdfPdf         = bsdfPdf;
            prd->woPdf           = woPdf;
            prd->bsdfVal         = bsdfVal;
            prd->throughPut     *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine          = cosine;
            setRayDirection(newDirection);
        }
        else {
            const auto reflCos   = rtlib::max(rtlib::dot(reflectDir, newDirection3), 0.0f);
            prd->bsdfPdf         = (shinness + 2.0f) * powf(reflCos, shinness) / RTLIB_M_2PI;
            prd->dTreePdf        = 0.0f;
            prd->woPdf           = prd->bsdfPdf;
            prd->bsdfVal         = (specular * prd->bsdfPdf / a_specular);
            prd->throughPut     *= (specular * rtlib::max(cosine3, 0.0f) / a_specular);
            prd->cosine          = cosine3;
            setRayDirection(newDirection3);
        }
    }
    else {
        //printf("Hit!\n");
        //反射しない
        prd->bsdfVal  = make_float3(1.0f);
        prd->woPdf    = 0.0f;
        prd->bsdfPdf  = 0.0f;
        prd->dTreePdf = 0.0f;
        prd->cosine   = 0.0f;
        prd->throughPut = make_float3(0.0f);
        prd->dTree    = nullptr;
        prd->done     = true;
    }
    prd->countEmitted = true;
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_phong_pg_nee() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const auto rayDirection = optixGetWorldRayDirection();
    const auto primitiveID = optixGetPrimitiveIndex();

    const auto v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const auto v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const auto v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const auto n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const auto normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const auto barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];


    const auto reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);

    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const auto emission = hgData->getEmissionColor(texCoord);
    const auto diffuse = hgData->getDiffuseColor(texCoord);
    const auto specular = hgData->getSpecularColor(texCoord);
    const auto shinness = hgData->shinness;
    const auto distance = optixGetRayTmax();
    const auto position = optixGetWorldRayOrigin() + distance * rayDirection;
    //direction
    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float3 newDirection3 = make_float3(0.0f);
    //cosine
    float  cosine1 = 0.0f;
    float  cosine2 = 0.0f;
    float  cosine3 = 0.0f;
    //payLoad
    RadiancePRD* prd    = getRadiancePRD();
    auto dTreeVoxelSize = make_float3(0.0f);
    const auto dTree    = params.sdTree.GetDTreeWrapper(position, dTreeVoxelSize);
    auto prvThroughPut  = prd->throughPut;
    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->radiance       = emission * prvThroughPut * static_cast<float>(prd->countEmitted);
    prd->distance       = distance;
    prd->isDelta        = false;
    if (prd->done) {
        return;
    }
    rtlib::Xorshift32 xor32(prd->seed);
    //const auto isValid = false;
    setRayOrigin(position);
    if (params.isBuilt) {
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(normal, newDirection1);
    }
    {
        rtlib::ONB onb(normal);
        newDirection2 = onb.local(rtlib::random_cosine_direction(xor32));
        cosine2 = rtlib::dot(normal, newDirection2);

    }
    {
        rtlib::ONB onb(reflectDir);
        const auto cosTht = powf(rtlib::random_float1(0.0f, 1.0f, xor32), 1.0f / (shinness + 1.0f));
        const auto sinTht = sqrtf(1.0f - cosTht * cosTht);
        const auto phi = rtlib::random_float1(0.0f, RTLIB_M_2PI, xor32);
        newDirection3 = onb.local(make_float3(sinTht * cosf(phi), sinTht * sinf(phi), cosTht));
        cosine3 = rtlib::dot(normal, newDirection3);
    }
    const auto  a_diffuse = (diffuse.x + diffuse.y + diffuse.z) / 3.0f;
    const auto  a_specular = (specular.x + specular.y + specular.z) / 3.0f;
    const float rnd1 = rtlib::random_float1(xor32);
    const float rnd2 = rtlib::random_float1(xor32);

    if (rnd1 < a_diffuse) {
        const auto  newDirection = rnd2 < 0.5f ? newDirection1 : newDirection2;
        const auto  cosine = rnd2 < 0.5f ? cosine1 : cosine2;
        const auto  bsdfPdf = rtlib::max(cosine / RTLIB_M_PI, 0.0f);
        const auto  bsdfVal = diffuse / (RTLIB_M_PI * a_diffuse);
        //両方とも正なら
        if (params.isBuilt) {
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf = 0.5f * dTreePdf + 0.5f * bsdfPdf;
            prd->bsdfVal = bsdfVal;
            prd->dTreePdf = dTreePdf;
            prd->bsdfPdf = bsdfPdf;
            prd->woPdf = woPdf;
            prd->throughPut *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine = cosine;
            setRayDirection(newDirection);
        }
        else {
            prd->bsdfVal = (diffuse / (RTLIB_M_PI * a_diffuse));
            prd->bsdfPdf = fabsf(cosine2) / RTLIB_M_PI;
            prd->dTreePdf = 0.0f;
            prd->woPdf = prd->bsdfPdf;
            prd->throughPut *= (diffuse / a_diffuse);
            prd->cosine = cosine2;
            setRayDirection(newDirection2);
        }
        {
            const float2 z = rtlib::random_float2(xor32);
            const auto   light = params.light;
            const float3 lightPos = light.corner + light.v1 * z.x + light.v2 * z.y;
            const float  Ldist = rtlib::distance(lightPos, position);
            const float3 lightDir = rtlib::normalize(lightPos - position);
            const float  ndl = rtlib::dot(normal, lightDir);
            const float  lndl = -rtlib::dot(light.normal, lightDir);
            const auto  diffuseLobe = diffuse / (a_diffuse*RTLIB_M_PI);
            float weight = 0.0f;
            if (ndl > 0.0f && lndl > 0.0f) {
                const bool occluded = traceOccluded(params.gasHandle, position, lightDir, 0.01f, Ldist - 0.01f);
                if (!occluded) {
                    //printf("not Occluded!\n");
                    const float A = rtlib::length(rtlib::cross(light.v1, light.v2));
                    weight = ndl * lndl * A / (Ldist * Ldist);
                }
            }
            prd->radiance += light.emission * prvThroughPut * weight * diffuseLobe;
        }
        prd->countEmitted = false;
    }
    else if (rnd1 < a_diffuse + a_specular)
    {
        const auto  newDirection = rnd2 < 0.5f ? newDirection1 : newDirection3;
        const auto  cosine = rnd2 < 0.5f ? cosine1 : cosine3;
        const auto  bsdfPdf = (shinness + 2.0f) * powf(rtlib::max(rtlib::dot(reflectDir, newDirection), 0.0f), shinness) / RTLIB_M_2PI;
        const auto  bsdfVal = specular * bsdfPdf / a_specular;
        //両方とも正なら
        if (params.isBuilt)
        {
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf = 0.5f * dTreePdf + 0.5f * bsdfPdf;
            //printf("Hit1! %f %f\n", woPdf,dTreePdf);
            prd->dTreePdf = dTreePdf;
            prd->bsdfPdf = bsdfPdf;
            prd->woPdf = woPdf;
            prd->bsdfVal = bsdfVal;
            prd->throughPut *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine = cosine;
            setRayDirection(newDirection);
        }
        else {
            const auto reflCos = rtlib::max(rtlib::dot(reflectDir, newDirection3), 0.0f);
            prd->bsdfPdf = (shinness + 2.0f) * powf(reflCos, shinness) / RTLIB_M_2PI;
            prd->dTreePdf = 0.0f;
            prd->woPdf = prd->bsdfPdf;
            prd->bsdfVal = (specular * prd->bsdfPdf / a_specular);
            prd->throughPut *= (specular * rtlib::max(cosine3, 0.0f) / a_specular);
            prd->cosine = cosine3;
            setRayDirection(newDirection3);
        }
        prd->countEmitted = true;
    }
    else {
        //printf("Hit!\n");
        //反射しない
        prd->bsdfVal = make_float3(1.0f);
        prd->woPdf = 0.0f;
        prd->bsdfPdf = 0.0f;
        prd->dTreePdf = 0.0f;
        prd->cosine = 0.0f;
        prd->throughPut = make_float3(0.0f);
        prd->dTree = nullptr;
        prd->done = true;
    }
    prd->seed = xor32.m_seed;
}
