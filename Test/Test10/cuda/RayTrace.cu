#include "hip/hip_runtime.h"
#define __HIPCC__
#include "RayTrace.h"
extern "C" {
    __constant__ Params params;
}
static __forceinline__ void trace(OptixTraversableHandle handle,const float3& rayOrigin, const float3& rayDirection,float tmin, float tmax,float3& color) {
    unsigned int p0, p1,p2;
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 1, 0, p0, p1, p2);
    color.x = int_as_float(p0);
    color.y = int_as_float(p1);
    color.z = int_as_float(p2);
}
extern "C" __global__ void     __raygen__rg(){
    const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();
    auto* rgData    = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u  = rgData->u;
	const float3 v  = rgData->v;
	const float3 w  = rgData->w;
	const float2 d  = make_float2(
		(2.0f * static_cast<float>(idx.x)/static_cast<float>(dim.x)) - 1.0,
		(2.0f * static_cast<float>(idx.y)/static_cast<float>(dim.y)) - 1.0);
	const float3 origin    = rgData->eye;
	const float3 direction = rtlib::normalize(d.x * u + d.y * v + w);
    //printf("%f, %lf, %lf\n", direction.x, direction.y, direction.z);
    float3 color;
    trace(params.gasHandle, origin,direction, 0.0f, 1e16f,color);
   // printf("%f, %lf\n", texCoord.x, texCoord.y);
    params.image[params.width * idx.y + idx.x] = make_uchar4(static_cast<unsigned char>(255.99 * color.x), static_cast<unsigned char>(255.99 * color.y), static_cast<unsigned char>(255.99 * color.z), 255);
}
extern "C" __global__ void       __miss__ms(){
    auto* msData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    optixSetPayload_0(float_as_int(msData->bgColor.x));
    optixSetPayload_1(float_as_int(msData->bgColor.y));
    optixSetPayload_2(float_as_int(msData->bgColor.z));
}
extern "C" __global__ void __closesthit__ch(){
    auto* hgData     = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    float2 texCoord  = optixGetTriangleBarycentrics();
    auto primitiveId = optixGetPrimitiveIndex();
    //printf("%d\n", primitiveId);
    auto p0          = hgData->vertices[hgData->indices[primitiveId].x];
    auto p1          = hgData->vertices[hgData->indices[primitiveId].y];
    auto p2          = hgData->vertices[hgData->indices[primitiveId].z];
    auto normal      = rtlib::normalize(rtlib::cross(p1 - p0, p2 - p0));
    auto diffTex = hgData->diffuseTex;
    auto t0      = hgData->texCoords[hgData->indices[primitiveId].x];
    auto t1      = hgData->texCoords[hgData->indices[primitiveId].y];
    auto t2      = hgData->texCoords[hgData->indices[primitiveId].z];
    auto t       = (1.0f-texCoord.x-texCoord.y)*t0 + texCoord.x * t1 + texCoord.y * t2;
    auto diffC   = tex2D<uchar4>(diffTex, t.x, t.y);
    //printf("%f %f\n",t0.x,t0.y);
    //optixSetPayload_0(float_as_int(float(diffC.x) / 255.99f));
    //optixSetPayload_1(float_as_int(float(diffC.y) / 255.99f));
    //optixSetPayload_2(float_as_int(float(diffC.z) / 255.99f));
    //optixSetPayload_0(float_as_int((t.x)));
    //optixSetPayload_1(float_as_int((t.y)));
    //optixSetPayload_2(float_as_int((2.0f-t.x-t.y)/2.0f));
    optixSetPayload_0(float_as_int((0.5f+0.5f*normal.x)));
    optixSetPayload_1(float_as_int((0.5f+0.5f*normal.y)));
    optixSetPayload_2(float_as_int((0.5f+0.5f*normal.z)));
}
extern "C" __global__ void     __anyhit__ah(){
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
}
