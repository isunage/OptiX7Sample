#include "hip/hip_runtime.h"
#define __HIPCC__
#include <hip/hip_runtime.h>
#include <RayTrace.h>
using namespace test24_restir;
extern "C" __global__ void combineSpatialReservoirs(
    Reservoir<LightRec> * inResvBuffer,
    Reservoir<LightRec> * outResvBuffer,
    ReservoirState      * tmpStatBuffer,
    RaySecondParams     * params, 
    int                   width, 
    int                   height, 
    int                   sample,
    int                   range) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < width && j < height) {
        auto origin    = params->posiBuffer[width * j + i];
        auto normal    = params->normBuffer[width * j + i];
        auto diffuse   = params->diffBuffer[width * j + i];
        auto seed      = params->seedBuffer[width * j + i];
        auto distance  = params->distBuffer[width * j + i];
        auto xor32     = rtlib::Xorshift32(seed);
        Reservoir<LightRec> r;
        float p_q = tmpStatBuffer[width * j + i].targetDensity;
        //First: Combine CurResv
        Reservoir<LightRec> curResv = inResvBuffer[width * j + i];
        {
            if (curResv.w_sum <= 0.0f) {
                p_q           = 0.0f;
                curResv.w_sum = 0.0f;
            }
            r.Update(curResv.y, p_q * curResv.w * static_cast<float>(curResv.m), rtlib::random_float1(xor32));
            r.m = curResv.m;
        }
        //Second: Combine NearResv
        for (int k = 0; k < sample; ++k)
        {
            /**/
            int s = i + cosf(rtlib::random_float1(xor32) * RTLIB_M_2PI) * static_cast<float>(range);
            int t = j + sinf(rtlib::random_float1(xor32) * RTLIB_M_2PI) * static_cast<float>(range);
            if (s<0 || s > width - 1 || t<0 || t> height-1||((s==i)&&(t==j))) {
                continue;
            }
            /**/
            float3 near_normal   = params->normBuffer[t * width + s];
            float  near_distance = params->distBuffer[t * width + s];
            if (rtlib::dot(near_normal, normal) < 0.90f ||
                fabsf((near_distance - distance)/distance) > 0.10f) {
                continue;
            }

            auto r_i   = inResvBuffer[width * t + s];
            
            float3 ldir  = r_i.y.position - origin;
            //Distance
            float  ldist = rtlib::length(ldir);
                   ldir /= static_cast<float>(ldist);
            //Bsdf
            float3 bsdf  = diffuse * RTLIB_M_INV_PI;
            //Emission
            float3 l_e   = r_i.y.emission;
            //Geometry 
            float  g     = fabsf(rtlib::dot(normal, ldir)) * fabsf(rtlib::dot(r_i.y.normal, ldir)) / (ldist * ldist);
            //Indirect Illumination
            float3 lp    = bsdf * l_e * g;
            float  lp_q  = (lp.x + lp.y + lp.z) / 3.0f;
            if (r.Update(r_i.y, lp_q * r_i.w * static_cast<float>(r_i.m), rtlib::random_float1(xor32))) {
                p_q = lp_q;
            }
            r.m += r_i.m;
        }
        r.w  = (p_q <= 0.0f) ? 0.0f : (r.w_sum / (static_cast<float>(r.m) * p_q));
        outResvBuffer[width * j + i]               = r;
        tmpStatBuffer[width * j + i].targetDensity = p_q;
        params->seedBuffer[width * j + i]          = xor32.m_seed;
    }
}

extern "C" __global__ void combineTemporalReservoirs(
    Reservoir<LightRec> * prvResvBuffer,
    Reservoir<LightRec> * curResvBuffer,
    ReservoirState      * tmpStatBuffer,
    RaySecondParams     * params, 
    int                   width, 
    int                   height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= width || j >= height) {
        return;
    }
    auto dIdx = params->motiBuffer[width * j + i];
    int  s = i + dIdx.x;
    int  t = j + dIdx.y;
    if (s<0 || s >= width || t< 0 || t >= height) {
        return;
    }
    auto curNormal   = params->normBuffer[width * j + i];
    auto curDistance = params->distBuffer[width * j + i];
    auto prvNormal   = params->normBuffer[width * t + s];
    auto prvDistance = params->distBuffer[width * t + s];
    if (rtlib::dot(curNormal, prvNormal) < 0.90f ||
        fabsf((prvDistance - curDistance) / curDistance) > 0.10f) {
        return;
    }

    auto origin     = params->posiBuffer[width * j + i];
    auto curDiffuse = params->diffBuffer[width * j + i];
    auto seed       = params->seedBuffer[width * j + i];
    auto xor32      = rtlib::Xorshift32(seed);
    Reservoir<LightRec> r;
    float p_q = tmpStatBuffer[width * j + i].targetDensity;
    //First: Combine CurResv
    {
        Reservoir<LightRec> curResv = curResvBuffer[width * j + i];
        if (curResv.w_sum <= 0.0f) {
            p_q = 0.0f;
            curResv.w_sum = 0.0f;
        }
        r.Update(curResv.y, p_q * curResv.w * static_cast<float>(curResv.m), rtlib::random_float1(xor32));
        r.m = curResv.m;
    }
    {
        auto prvResv   = prvResvBuffer[width * t + s];
        prvResv.m      = rtlib::min(prvResv.m, 20 * r.m);
        //selective probability on current pixel
        float3   ldir  = prvResv.y.position - origin;
        float    ldist = rtlib::length(ldir);
        ldir /= static_cast<float>(ldist);
        float3   bsdf = curDiffuse * RTLIB_M_INV_PI;
        float3   le   = prvResv.y.emission;
        float    g    = fabs(rtlib::dot(ldir, curNormal)) * fabs(rtlib::dot(ldir, prvResv.y.normal)) / (ldist * ldist);
        float3   lp   = bsdf * le * g;
        float    lp_q = (lp.x + lp.y + lp.z) / 3.0f;
        if (r.Update(prvResv.y, lp_q * prvResv.w * static_cast<float>(prvResv.m), rtlib::random_float1(xor32))) {
            p_q = lp_q;
        }
        r.m += prvResv.m;
    }
    r.w = (p_q <= 0.0f) ? 0.0f : (r.w_sum / (static_cast<float>(r.m) * p_q));
    curResvBuffer[width * j + i] = r;
    tmpStatBuffer[width * j + i].targetDensity = p_q;
    params->seedBuffer[width * j + i] = xor32.m_seed;
}