#include "hip/hip_runtime.h"
#define __HIPCC__
#include "RayTrace.h"
#include "PathGuiding.h"
using namespace test24_restir_guide;
struct RadiancePRD {
    DTreeWrapper* dTree;
    float3        dTreeVoxelSize;
    float3        radiance;
    float3        bsdfVal;
    float3        throughPut;
    float         woPdf, bsdfPdf, dTreePdf;
    float         cosine;
    float         distance;
    unsigned int  seed;
    bool          isDelta;
    bool          countEmitted;
    bool          done;
};
extern "C" {
    __constant__ RayTraceParams params;
}
template<typename RNG>
static __forceinline__ __device__ float3       sampleCosinePDF(const float3& normal, RNG& rng)
{
    rtlib::ONB onb(normal);
    return onb.local(rtlib::random_cosine_direction(rng));
}
template<typename RNG>
static __forceinline__ __device__ float3       samplePhongPDF(const float3& reflectDir, float shinness, RNG& rng)
{
    rtlib::ONB onb(reflectDir);
    const auto cosTht = powf(rtlib::random_float1(0.0f, 1.0f, rng), 1.0f / (shinness + 1.0f));
    const auto sinTht = sqrtf(1.0f - cosTht * cosTht);
    const auto phi    = rtlib::random_float1(0.0f, RTLIB_M_2PI, rng);
    return onb.local(make_float3(sinTht * cosf(phi), sinTht * sinf(phi), cosTht));
}
static __forceinline__ __device__ float        getValPhongPDF(const float3& direction, const float3& reflectDir, float shinness)
{

    const auto reflCos = rtlib::max(rtlib::dot(reflectDir, direction), 0.0f);
    return (shinness + 2.0f) * powf(reflCos, shinness) / RTLIB_M_2PI;
}
static __forceinline__ __device__ float3       faceForward(const float3& n, const float3& i, const float3& nref) {
    return copysignf(1.0f, rtlib::dot(n, i)) * nref;
}
static __forceinline__ __device__ void*        unpackPointer(unsigned int p0, unsigned int p1) {
    return reinterpret_cast<void*>(rtlib::to_combine(p0, p1));
}
static __forceinline__ __device__ void         packPointer(void* ptr, unsigned int& p0, unsigned int& p1) {
    const unsigned long long llv = reinterpret_cast<const unsigned long long>(ptr);
    p0 = rtlib::to_upper(llv);
    p1 = rtlib::to_lower(llv);
}
static __forceinline__ __device__ float3       unpackFloat3(unsigned int p0, unsigned p1, unsigned int p2)
{
    return make_float3(__uint_as_float(p0), __uint_as_float(p1), __uint_as_float(p2));
}
static __forceinline__ __device__ void         packFloat3(const float3& v, unsigned int& p0, unsigned& p1, unsigned int& p2)
{
    p0 = __float_as_uint(v.x);
    p1 = __float_as_uint(v.y);
    p2 = __float_as_uint(v.z);
}
static __forceinline__ __device__ SurfaceRec   getSurfaceRec(const SurfaceParameters& surfParams)
{
    SurfaceRec srec;
    srec.distance = optixGetRayTmax();
    srec.position = optixGetWorldRayOrigin() + srec.distance * optixGetWorldRayDirection();
    const int    primitiveID  = optixGetPrimitiveIndex();
    const float2 barycentric  = optixGetTriangleBarycentrics();
    const uint3  triIndex     = surfParams.indices[primitiveID];
    const float3 v0 = surfParams.vertices[triIndex.x];
    const float3 v1 = surfParams.vertices[triIndex.y];
    const float3 v2 = surfParams.vertices[triIndex.z];
    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    srec.vNormal = n0;
    srec.sNormal = n0;
    if (surfParams.normals) {
        const float3 nv0 = surfParams.normals[triIndex.x];
        const float3 nv1 = surfParams.normals[triIndex.y];
        const float3 nv2 = surfParams.normals[triIndex.z];
        const bool isValidNv0 = !((nv0.x == 0.0f) && (nv0.y == 0.0f) && (nv0.z == 0.0f));
        const bool isValidNv1 = !((nv1.x == 0.0f) && (nv1.y == 0.0f) && (nv1.z == 0.0f));
        const bool isValidNv2 = !((nv2.x == 0.0f) && (nv2.y == 0.0f) && (nv2.z == 0.0f));
        if (isValidNv0 && isValidNv1 && isValidNv2)
        {
            float3 nv = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize((1.0f - barycentric.x - barycentric.y) * nv0 + barycentric.x * nv1 + barycentric.y * nv2));
            if (rtlib::dot(nv, n0) > 0.0f) {
                srec.vNormal = nv;
            }
        }
    }
    const auto t0 = surfParams.texCoords[triIndex.x];
    const auto t1 = surfParams.texCoords[triIndex.y];
    const auto t2 = surfParams.texCoords[triIndex.z];
    srec.texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    return srec;
}
static __forceinline__ __device__ RadiancePRD* getRadiancePRD() {
    unsigned int p0 = optixGetPayload_0();
    unsigned int p1 = optixGetPayload_1();
    return static_cast<RadiancePRD*>(unpackPointer(p0, p1));
}
static __forceinline__ __device__ void         setRadiancePRD(RadiancePRD* prd) {
    unsigned int p0;
    unsigned int p1;
    packPointer(static_cast<void*>(prd), p0, p1);
    optixSetPayload_0(p0);
    optixSetPayload_1(p1);
}
static __forceinline__ __device__ void         setRayOrigin(const float3& origin) {
    unsigned int p2, p3, p4;
    packFloat3(origin, p2, p3, p4);
    optixSetPayload_2(p2);
    optixSetPayload_3(p3);
    optixSetPayload_4(p4);
}
static __forceinline__ __device__ void         setRayDirection(const float3& direction) {
    unsigned int p5, p6, p7;
    packFloat3(direction, p5, p6, p7);
    optixSetPayload_5(p5);
    optixSetPayload_6(p6);
    optixSetPayload_7(p7);
}
static __forceinline__ __device__ void         setPayloadOccluded(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}
static __forceinline__ __device__ void         traceRadiance(OptixTraversableHandle handle,float3& rayOrigin,float3& rayDirection,float tmin, float tmax, RadiancePRD* prd) {
    unsigned int p0, p1, p2, p3, p4, p5, p6, p7;
    packPointer(prd, p0, p1);
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, RAY_TYPE_RADIANCE, RAY_TYPE_COUNT, RAY_TYPE_RADIANCE, p0, p1, p2, p3, p4, p5, p6, p7);
    rayOrigin    = unpackFloat3(p2, p3, p4);
    rayDirection = unpackFloat3(p5, p6, p7);
}
static __forceinline__ __device__ bool         traceOccluded(OptixTraversableHandle handle,const float3& rayOrigin,const float3& rayDirection,float tmin, float tmax) {
    unsigned int occluded = false;
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, RAY_TYPE_OCCLUSION, RAY_TYPE_COUNT, RAY_TYPE_OCCLUSION, occluded);
    return occluded;
}
extern "C" __global__ void __raygen__def() {

    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    auto* rgData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u = rgData->u;
    const float3 v = rgData->v;
    const float3 w = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.radiance = make_float3(0.0f);
        prd.bsdfVal  = make_float3(1.0f);
        prd.throughPut = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.cosine       = 0.0f;
        prd.distance     = 0.0f;
        prd.done         = false;
        prd.countEmitted = true;
        prd.isDelta      = false;
        prd.seed         = seed;
        int depth = 0;
        for (;;) {
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            //vertices�̍X�V
            //Radiance�̍X�V
            //Result�̍X�V
            result += prd.radiance;
            //ThroughPut�̍X�V
            bool isValidThroughPut = (!isnan(prd.throughPut.x) && !isnan(prd.throughPut.y) && !isnan(prd.throughPut.z) &&
                isfinite(prd.throughPut.x) && isfinite(prd.throughPut.y) && isfinite(prd.throughPut.z) &&
                (prd.throughPut.x >= 0.0f && prd.throughPut.y >= 0.0f && prd.throughPut.z >= 0.0f));
            bool isValidDirection  = (!isnan(rayDirection.x) && !isnan(rayDirection.y) && !isnan(rayDirection.z));
            if (prd.done || depth >= params.maxTraceDepth-1 || !isValidThroughPut || !isValidDirection) {
                break;
            }
            depth++;
        }
        seed = prd.seed;
    } while(--i);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor = prevAccumColor + result;
        float3 frameColor = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
        params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __raygen__pg_def() {

    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    auto* rgData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u = rgData->u;
    const float3 v = rgData->v;
    const float3 w = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    TraceVertex vertices[RAY_TRACE_MAX_VERTEX_COUNT] = {};
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.radiance    = make_float3(0.0f);
        prd.bsdfVal     = make_float3(1.0f);
        prd.throughPut  = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.distance     = 0.0f;
        prd.done         = false;
        prd.isDelta      = false;
        prd.countEmitted = true;
        prd.seed = seed;
        int depth = 0;
        for (;;) {
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            if(!params.isFinal){
                vertices[depth].rayOrigin      = rayOrigin;
                vertices[depth].rayDirection   = rayDirection;
                vertices[depth].dTree          = prd.dTree;
                vertices[depth].dTreeVoxelSize = prd.dTreeVoxelSize;
                vertices[depth].throughPut     = prd.throughPut;
                vertices[depth].bsdfVal        = prd.bsdfVal;
                vertices[depth].radiance       = make_float3(0.0f);
                vertices[depth].woPdf          = prd.woPdf;
                vertices[depth].bsdfPdf        = prd.bsdfPdf;
                vertices[depth].dTreePdf       = prd.dTreePdf;
                vertices[depth].cosine         = prd.cosine;
                vertices[depth].isDelta        = prd.isDelta;
                for (int j = 0; j < depth; ++j) {
                    vertices[j].Record(prd.radiance);
                }
            }
            //OK
            result += prd.radiance;
            bool isValidThroughPut = (!isnan(prd.throughPut.x) && !isnan(prd.throughPut.y) && !isnan(prd.throughPut.z) &&
                                    isfinite(prd.throughPut.x) && isfinite(prd.throughPut.y) && isfinite(prd.throughPut.z) &&
                                            (prd.throughPut.x >= 0.0f && prd.throughPut.y >= 0.0f && prd.throughPut.z >= 0.0f));
            bool isValidDirection  = (!isnan(rayDirection.x) && !isnan(rayDirection.y) && !isnan(rayDirection.z));
            if (prd.done || depth >= params.maxTraceDepth - 1 || !isValidThroughPut || !isValidDirection) {
                break;
            }
            depth++;
        }
        if(!params.isFinal){
            for (int j = 0; j < depth; ++j) {
                vertices[j].Commit<RAY_TRACE_S_FILTER, RAY_TRACE_D_FILTER>(params.sdTree, 1.0f);
            }
        }
        seed = prd.seed;
    } while(--i);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor     = prevAccumColor + result;
        float3 frameColor           = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor                  = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
        params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __raygen__pg_nee() {
    const uint3 idx   = optixGetLaunchIndex();
    const uint3 dim   = optixGetLaunchDimensions();
    auto* rgData      = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u    = rgData->u;
    const float3 v    = rgData->v;
    const float3 w    = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    TraceVertex vertices[RAY_TRACE_MAX_VERTEX_COUNT] = {};
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.radiance = make_float3(0.0f);
        prd.bsdfVal = make_float3(1.0f);
        prd.throughPut = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.distance = 0.0f;
        prd.done = false;
        prd.isDelta = false;
        prd.countEmitted = true;
        prd.seed = seed;
        int depth = 0;
        for (;;) {
            if (depth >= params.maxTraceDepth-1) {
                prd.done = true;
            }
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            if (!params.isFinal) {
                vertices[depth].rayOrigin    = rayOrigin;
                vertices[depth].rayDirection = rayDirection;
                vertices[depth].dTree = prd.dTree;
                vertices[depth].dTreeVoxelSize = prd.dTreeVoxelSize;
                vertices[depth].throughPut = prd.throughPut;
                vertices[depth].bsdfVal = prd.bsdfVal;
                vertices[depth].radiance = make_float3(0.0f);
                vertices[depth].woPdf = prd.woPdf;
                vertices[depth].bsdfPdf = prd.bsdfPdf;
                vertices[depth].dTreePdf = prd.dTreePdf;
                vertices[depth].cosine = prd.cosine;
                vertices[depth].isDelta = prd.isDelta;
                for (int j = 0; j < depth; ++j) {
                    vertices[j].Record(prd.radiance);
                }
            }
            //OK
            //Result�̍X�V
            result += prd.radiance;

            bool isValidThroughPut = (!isnan(prd.throughPut.x) && !isnan(prd.throughPut.y)   && !isnan(prd.throughPut.z)   &&
                                    isfinite(prd.throughPut.x) && isfinite(prd.throughPut.y) && isfinite(prd.throughPut.z) &&
                                     (prd.throughPut.x >= 0.0f &&   prd.throughPut.y >= 0.0f && prd.throughPut.z >= 0.0f));
            bool isValidDirection  = (!isnan(rayDirection.x)   && !isnan(rayDirection.y)     && !isnan(rayDirection.z));

            if (prd.done || !isValidThroughPut || !isValidDirection) {
                break;
            }
            depth++;
        }
        if (!params.isFinal) {
            for (int j = 0; j < depth; ++j) {
                vertices[j].Commit<RAY_TRACE_S_FILTER, RAY_TRACE_D_FILTER>(params.sdTree, 1.0f);
            }
        }
        seed = prd.seed;
    } while (--i);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor = prevAccumColor + result;
        float3 frameColor = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
        params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __miss__radiance() {
    auto* msData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    RadiancePRD* prd = getRadiancePRD();
    prd->radiance = make_float3(msData->bgColor.x, msData->bgColor.y, msData->bgColor.z) * prd->throughPut;
    prd->dTree    = nullptr;
    prd->woPdf    = 1.0f;
    prd->bsdfPdf  = 0.0f;
    prd->dTreePdf = 0.0f;
    prd->bsdfVal  = make_float3(1.0f);
    prd->cosine   = 0.0f;
    prd->distance = optixGetRayTmax();
    prd->done     = true;
}
extern "C" __global__ void __miss__occluded() {
    setPayloadOccluded(false);
}
//diffuse
extern "C" __global__ void __closesthit__radiance_for_diffuse_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec = hgData->matParams.GetRecord(srec.texCoord);

    RadiancePRD* prd = getRadiancePRD();
    float3 newDirection   = make_float3(0.0f);

    prd->dTree    = nullptr;
    prd->radiance = mrec.emission * prd->throughPut;
    prd->distance = srec.distance;

    rtlib::Xorshift32 xor32(prd->seed);

    newDirection      = sampleCosinePDF(srec.sNormal,xor32);
    const auto cosine = rtlib::dot(newDirection, srec.sNormal);

    prd->bsdfPdf  = fabsf(cosine) / RTLIB_M_PI;
    prd->dTreePdf = 0.0f;
    prd->woPdf    = prd->bsdfPdf;

    setRayOrigin(srec.position);
    setRayDirection(newDirection);

    prd->cosine      = cosine;
    prd->bsdfVal     = mrec.diffuse / RTLIB_M_PI;
    prd->throughPut *= mrec.diffuse;
    prd->seed        = xor32.m_seed;

    prd->countEmitted= true;
    prd->isDelta     = false;
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_nee() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec = hgData->matParams.GetRecord(srec.texCoord);
    RadiancePRD* prd = getRadiancePRD();

    float3 newDirection = make_float3(0.0f);
    const auto prvThroughPut = prd->throughPut;
    prd->dTree = nullptr;
    prd->radiance = mrec.emission * prvThroughPut * static_cast<float>(prd->countEmitted);
    prd->distance = srec.distance;
    if (prd->done) {
        return;
    }
    rtlib::Xorshift32 xor32(prd->seed);
    rtlib::ONB onb(srec.sNormal);
    newDirection = onb.local(rtlib::random_cosine_direction(xor32));

    const auto cosine = rtlib::dot(newDirection, srec.sNormal);

    prd->bsdfPdf = fabsf(cosine) / RTLIB_M_PI;
    prd->dTreePdf = 0.0f;
    prd->woPdf = prd->bsdfPdf;

    setRayOrigin(srec.position);
    setRayDirection(newDirection);

    prd->cosine = cosine;
    prd->bsdfVal = mrec.diffuse/ RTLIB_M_PI;
    prd->throughPut *= mrec.diffuse;

    prd->countEmitted = false;
    prd->isDelta = false;

    {
        LightRec lRec = {};
        auto  distance = 0.0f;
        auto  invAreaP = 0.0f;
        auto  lightDir = params.light.Sample(srec.position, lRec, distance, invAreaP, xor32);
        auto  ndl      = rtlib::dot(lightDir, srec.sNormal);
        auto  emission = lRec.emission;
        auto  bsdf     = mrec.diffuse / RTLIB_M_PI;
        auto  g        = ndl * fabsf(rtlib::dot(lightDir, lRec.normal)) /(distance * distance);
        auto  f        = emission * bsdf * g;
        auto  f_a      = (f.x + f.y + f.z) / 3.0f;
        auto  weight   = make_float3(0.0f);
        if (f_a > 0.0f && invAreaP > 0.0f) {
            const bool occluded = traceOccluded(params.gasHandle, srec.position, lightDir, 0.01f, distance - 0.01f);
            if (!occluded) {
                weight = f * invAreaP;
            }
        }
        prd->radiance += prvThroughPut * weight;
    }

    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_pg_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec = hgData->matParams.GetRecord(srec.texCoord);
    RadiancePRD* prd = getRadiancePRD();

    auto  dTreeVoxelSize= make_float3(0.0f);
    const auto dTree    = params.sdTree.GetDTreeWrapper(srec.position, dTreeVoxelSize);

    float3 newDirection1= make_float3(0.0f);
    float3 newDirection2= make_float3(0.0f);
    float  cosine1      = 0.0f;
    float  cosine2      = 0.0f;

    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->radiance       = mrec.emission * prd->throughPut;
    prd->distance       = srec.distance;
    prd->bsdfPdf        = 0.0f;
    prd->bsdfVal        = mrec.diffuse / RTLIB_M_PI;
    prd->isDelta        = false;
    rtlib::Xorshift32 xor32(prd->seed);

    setRayOrigin(srec.position);
    if(params.isBuilt){
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(srec.sNormal, newDirection1);
    }
    {
        newDirection2 = sampleCosinePDF(srec.sNormal,xor32);
        cosine2 = rtlib::dot(srec.sNormal, newDirection2);
    }

    const float rnd          = rtlib::random_float1(xor32);
    const auto  newDirection = rnd < params.sdTree.fraction ? newDirection1 : newDirection2;
    const auto  cosine       = rnd < params.sdTree.fraction ? cosine1 : cosine2;
    const auto  bsdfPdf      = rtlib::max(cosine / RTLIB_M_PI, 0.0f);
    //両方とも正なら
    if (params.isBuilt) {
        const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
        const auto  woPdf    = params.sdTree.fraction * bsdfPdf + (1.0f- params.sdTree.fraction) * dTreePdf;
        prd->bsdfPdf         = bsdfPdf;
        prd->dTreePdf        = dTreePdf;
        prd->woPdf           = woPdf;
        prd->throughPut     *=(prd->bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
        setRayDirection(newDirection);
        prd->cosine = cosine;
    }
    else {
        prd->bsdfPdf     = fabsf(cosine2) / RTLIB_M_PI;
        prd->dTreePdf    = 0.0f;
        prd->woPdf       = prd->bsdfPdf;
        prd->throughPut *= mrec.diffuse;
        setRayDirection(newDirection2);
        prd->cosine      = cosine2;
    }
    prd->countEmitted    = true;
    prd->seed            = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_pg_nee() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec = hgData->matParams.GetRecord(srec.texCoord);
    RadiancePRD* prd = getRadiancePRD();

    auto  dTreeVoxelSize = make_float3(0.0f);
    const auto dTree     = params.sdTree.GetDTreeWrapper(srec.position, dTreeVoxelSize);

    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float  cosine1 = 0.0f;
    float  cosine2 = 0.0f;

    auto prvThroughPut  = prd->throughPut;
    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    bool countEmitted   = prd->countEmitted || ((mrec.flags & MATERIAL_FLAG_USE_NEE_BIT) == 0);
    prd->radiance       = mrec.emission * prvThroughPut*static_cast<float>(countEmitted);
    prd->distance       = srec.distance;
    prd->bsdfPdf        = 0.0f;
    prd->bsdfVal        = mrec.diffuse / RTLIB_M_PI;
    prd->isDelta        = false;
    //new
    if (prd->done) {
        return;
    }
    rtlib::Xorshift32 xor32(prd->seed);
    setRayOrigin(srec.position);
    if (params.isBuilt) {
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(srec.sNormal, newDirection1);

        if (isnan(newDirection1.x) || isnan(newDirection1.y) || isnan(newDirection1.z)) {
            printf("newDirection1 is nan: new Direction1 = (%f, %f, %f) normal = (%f, %f, %f)\n", newDirection1.x, newDirection1.y, newDirection1.z, srec.sNormal.x, srec.sNormal.y, srec.sNormal.z);
        }
    }
    {
        newDirection2 = sampleCosinePDF(srec.sNormal, xor32);
        cosine2       = rtlib::dot(srec.sNormal, newDirection2);
        if (isnan(newDirection2.x) || isnan(newDirection2.y) || isnan(newDirection2.z))
        {
            printf("newDirection2 is nan!\n");
        }
    }

    const float rnd = rtlib::random_float1(xor32);
    const auto  newDirection = rnd < params.sdTree.fraction ? newDirection1 : newDirection2;
    const auto  cosine  = rnd < params.sdTree.fraction ? cosine1 : cosine2;
    const auto  bsdfPdf = rtlib::max(cosine / RTLIB_M_PI, 0.0f);
    //両方とも正なら
    if (params.isBuilt) {
        const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
        const auto  woPdf    = params.sdTree.fraction * bsdfPdf + (1.0f- params.sdTree.fraction) * dTreePdf;
        prd->bsdfPdf = bsdfPdf;
        prd->dTreePdf = dTreePdf;
        prd->woPdf = woPdf;
        prd->throughPut *= (prd->bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
        setRayDirection(newDirection);
        prd->cosine = cosine;
    }
    else {
        prd->bsdfPdf = fabsf(cosine2) / RTLIB_M_PI;
        prd->dTreePdf = 0.0f;
        prd->woPdf = prd->bsdfPdf;
        prd->throughPut *= mrec.diffuse;
        setRayDirection(newDirection2);
        prd->cosine = cosine2;
    }
    {
        Reservoir<LightRec> resv = {};
        auto f_y = make_float3(0.0f);
        auto f_a_y = 0.0f;
        auto lightDir_y = make_float3(0.0f);
        auto distance_y = 0.0f;
        for (int i = 0; i < params.numCandidates; ++i) {
            LightRec lRec = {};
            auto  distance = 0.0f;
            auto  invAreaP = 0.0f;
            auto  lightDir = params.light.Sample(srec.position, lRec, distance, invAreaP, xor32);
            auto  ndl = rtlib::dot(lightDir, srec.sNormal);
            auto  lndl = rtlib::dot(lightDir, lRec.normal);
            auto  emission = lRec.emission * static_cast<float>(lndl < 0.0f);
            auto  bsdf = mrec.diffuse * RTLIB_M_INV_PI;
            auto  g = ndl * fabsf(lndl)/ (distance * distance);
            auto  f = emission * bsdf * g;
            auto  f_a = (f.x + f.y + f.z) / 3.0f;
            if (resv.Update(lRec, f_a * invAreaP, rtlib::random_float1(xor32))) {
                f_y = f;
                f_a_y = f_a;
                lightDir_y = lightDir;
                distance_y = distance;
            }
        }
        float resv_w = 0.0f;
        if (resv.w_sum > 0.0f && f_a_y > 0.0f) {
            const bool occluded = traceOccluded(params.gasHandle, srec.position, lightDir_y, 0.01f, distance_y - 0.01f);
            resv_w = occluded ? 0.0f : (resv.w_sum / (f_a_y * static_cast<float>(resv.m)));
        }
        auto  weight = f_y * resv_w;
        prd->radiance += prvThroughPut * weight;
    }
    prd->countEmitted = false;
    prd->seed = xor32.m_seed;
}
//phong
extern "C" __global__ void __closesthit__radiance_for_phong_pg_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec = hgData->matParams.GetRecord(srec.texCoord);
    const auto reflectDir = rtlib::normalize(rtlib::reflect(rayDirection, srec.sNormal));
    //direction
    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float3 newDirection3 = make_float3(0.0f);
    //cosine
    float  cosine1 = 0.0f;
    float  cosine2 = 0.0f;
    float  cosine3 = 0.0f;
    //payLoad
    RadiancePRD* prd = getRadiancePRD();
    auto dTreeVoxelSize = make_float3(0.0f);
    const auto dTree = params.sdTree.GetDTreeWrapper(srec.position, dTreeVoxelSize);
    prd->dTree = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->radiance = mrec.emission * prd->throughPut;
    prd->distance = srec.distance;
    prd->isDelta = false;
    rtlib::Xorshift32 xor32(prd->seed);
    //const auto isValid = false;
    setRayOrigin(srec.position);
    if (params.isBuilt) {
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(srec.sNormal, newDirection1);
    }
    {
        newDirection2 = sampleCosinePDF(srec.sNormal, xor32);
        cosine2 = rtlib::dot(srec.sNormal, newDirection2);
        newDirection3 = samplePhongPDF(reflectDir, mrec.shinness, xor32);
        cosine3 = rtlib::dot(srec.sNormal, newDirection3);
    }

    const auto  a_diffuse = (mrec.diffuse.x + mrec.diffuse.y + mrec.diffuse.z) / 3.0f;
    const auto  a_specular = (mrec.specular.x + mrec.specular.y + mrec.specular.z) / 3.0f;
    const float rnd1 = rtlib::random_float1(xor32);

    if (rnd1 < a_diffuse) {
        //両方とも正なら
        if (params.isBuilt) {
            const float rnd2 = rtlib::random_float1(xor32);
            const auto  newDirection = rnd2 < params.sdTree.fraction ? newDirection1 : newDirection2;
            const auto  cosine = rnd2 < params.sdTree.fraction ? cosine1 : cosine2;
            const auto  bsdfPdf = rtlib::max(cosine / RTLIB_M_PI, 0.0f);
            const auto  bsdfVal = mrec.diffuse / (RTLIB_M_PI * a_diffuse);
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf = params.sdTree.fraction * dTreePdf + (1.0f - params.sdTree.fraction) * bsdfPdf;
            prd->bsdfVal = bsdfVal;
            prd->dTreePdf = dTreePdf;
            prd->bsdfPdf = bsdfPdf;
            prd->woPdf = woPdf;
            prd->throughPut *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine = cosine;
            setRayDirection(newDirection);
        }
        else {
            prd->bsdfVal = (mrec.diffuse / (RTLIB_M_PI * a_diffuse));
            prd->bsdfPdf = fabsf(cosine2) / RTLIB_M_PI;
            prd->dTreePdf = 0.0f;
            prd->woPdf = prd->bsdfPdf;
            prd->throughPut *= (mrec.diffuse / a_diffuse);
            prd->cosine = cosine2;
            setRayDirection(newDirection2);
        }
    }
    else if (rnd1 < a_diffuse + a_specular)
    {
        //両方とも正なら
        if (params.isBuilt)
        {
            const float rnd2 = rtlib::random_float1(xor32);
            const auto  newDirection = rnd2 < params.sdTree.fraction ? newDirection1 : newDirection3;
            const auto  cosine = rnd2 < params.sdTree.fraction ? cosine1 : cosine3;
            const auto  bsdfPdf = getValPhongPDF(newDirection, reflectDir, mrec.shinness);
            const auto  bsdfVal = mrec.specular * bsdfPdf / a_specular;
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf = params.sdTree.fraction * dTreePdf + (1.0f - params.sdTree.fraction) * bsdfPdf;
            //printf("Hit1! %f %f\n", woPdf,dTreePdf);
            prd->dTreePdf = dTreePdf;
            prd->bsdfPdf = bsdfPdf;
            prd->woPdf = woPdf;
            prd->bsdfVal = bsdfVal;
            prd->throughPut *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine = cosine;
            setRayDirection(newDirection);
        }
        else {
            prd->bsdfPdf = getValPhongPDF(newDirection3, reflectDir, mrec.shinness);
            prd->dTreePdf = 0.0f;
            prd->woPdf = prd->bsdfPdf;
            prd->bsdfVal = (mrec.specular * prd->bsdfPdf / a_specular);
            prd->throughPut *= (mrec.specular * rtlib::max(cosine3, 0.0f) / a_specular);
            prd->cosine = cosine3;
            setRayDirection(newDirection3);
        }
    }
    else {
        //printf("Hit!\n");
        //反射しない
        prd->bsdfVal = make_float3(1.0f);
        prd->woPdf = 0.0f;
        prd->bsdfPdf = 0.0f;
        prd->dTreePdf = 0.0f;
        prd->cosine = 0.0f;
        prd->throughPut = make_float3(0.0f);
        prd->dTree = nullptr;
        prd->done = true;
    }
    prd->countEmitted = true;
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_phong_pg_nee() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec = hgData->matParams.GetRecord(srec.texCoord);
    const auto reflectDir = rtlib::normalize(rtlib::reflect(rayDirection, srec.sNormal));

    RadiancePRD* prd = getRadiancePRD();
    //direction
    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float3 newDirection3 = make_float3(0.0f);
    //cosine
    float  cosine1 = 0.0f;
    float  cosine2 = 0.0f;
    float  cosine3 = 0.0f;
    //payLoad
    auto dTreeVoxelSize = make_float3(0.0f);
    const auto dTree = params.sdTree.GetDTreeWrapper(srec.position, dTreeVoxelSize);
    auto prvThroughPut = prd->throughPut;
    prd->dTree = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    bool countEmitted = prd->countEmitted || ((mrec.flags & MATERIAL_FLAG_USE_NEE_BIT) == 0);
    prd->radiance = mrec.emission * prvThroughPut * static_cast<float>(countEmitted);
    prd->distance = srec.distance;
    prd->isDelta = false;
    if (prd->done) {
        return;
    }
    rtlib::Xorshift32 xor32(prd->seed);
    //const auto isValid = false;
    setRayOrigin(srec.position);
    if (params.isBuilt) {
        newDirection1 = dTree->Sample(xor32);
        cosine1 = rtlib::dot(srec.sNormal, newDirection1);
    }
    {
        newDirection2 = sampleCosinePDF(srec.sNormal, xor32);
        cosine2 = rtlib::dot(srec.sNormal, newDirection2);
        newDirection3 = samplePhongPDF(reflectDir, mrec.shinness, xor32);
        cosine3 = rtlib::dot(srec.sNormal, newDirection3);
    }
    const auto  a_diffuse = (mrec.diffuse.x + mrec.diffuse.y + mrec.diffuse.z) / 3.0f;
    const auto  a_specular = (mrec.specular.x + mrec.specular.y + mrec.specular.z) / 3.0f;
    const float rnd1 = rtlib::random_float1(xor32);
    const float rnd2 = rtlib::random_float1(xor32);

    if (rnd1 < a_diffuse) {
        const auto  newDirection = rnd2 < params.sdTree.fraction ? newDirection1 : newDirection2;
        const auto  cosine = rnd2 < params.sdTree.fraction ? cosine1 : cosine2;
        const auto  bsdfPdf = rtlib::max(cosine / RTLIB_M_PI, 0.0f);
        const auto  bsdfVal = mrec.diffuse / (RTLIB_M_PI * a_diffuse);
        //両方とも正なら
        if (params.isBuilt) {
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf = params.sdTree.fraction * dTreePdf + (1.0f - params.sdTree.fraction) * bsdfPdf;
            prd->bsdfVal = bsdfVal;
            prd->dTreePdf = dTreePdf;
            prd->bsdfPdf = bsdfPdf;
            prd->woPdf = woPdf;
            prd->throughPut *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine = cosine;
            setRayDirection(newDirection);
        }
        else {
            prd->bsdfVal = (mrec.diffuse / (RTLIB_M_PI * a_diffuse));
            prd->bsdfPdf = fabsf(cosine2) / RTLIB_M_PI;
            prd->dTreePdf = 0.0f;
            prd->woPdf = prd->bsdfPdf;
            prd->throughPut *= (mrec.diffuse / a_diffuse);
            prd->cosine = cosine2;
            setRayDirection(newDirection2);
        }
        {
            const auto  diffuseLobe = mrec.diffuse / (a_diffuse * RTLIB_M_PI);
            Reservoir<LightRec> resv = {};
            auto f_y = make_float3(0.0f);
            auto f_a_y = 0.0f;
            auto lightDir_y = make_float3(0.0f);
            auto distance_y = 0.0f;
            for (int i = 0; i < params.numCandidates; ++i) {
                LightRec lRec = {};
                auto  distance = 0.0f;
                auto  invAreaP = 0.0f;
                auto  lightDir = params.light.Sample(srec.position, lRec, distance, invAreaP, xor32);
                auto  ndl  = rtlib::dot(lightDir, srec.sNormal);
                auto  lndl = rtlib::dot(lightDir, lRec.normal);
                auto  emission = lRec.emission * static_cast<float>(lndl <0.0f);
                auto  bsdf = diffuseLobe;
                auto  g = ndl * fabsf(lndl) / (distance * distance);
                auto  f = emission * bsdf * g;
                auto  f_a = (f.x + f.y + f.z) / 3.0f;
                if (resv.Update(lRec, f_a * invAreaP, rtlib::random_float1(xor32))) {
                    f_y = f;
                    f_a_y = f_a;
                    lightDir_y = lightDir;
                    distance_y = distance;
                }
            }
            float resv_w = 0.0f;
            if (resv.w_sum > 0.0f && f_a_y > 0.0f) {
                const bool occluded = traceOccluded(params.gasHandle, srec.position, lightDir_y, 0.01f, distance_y - 0.01f);
                resv_w = occluded ? 0.0f : (resv.w_sum / (f_a_y * static_cast<float>(resv.m)));
            }
            auto  weight = f_y * resv_w;
            prd->radiance += prvThroughPut * weight;
        }
        prd->countEmitted = false;
    }
    else if (rnd1 < a_diffuse + a_specular)
    {
        const auto  newDirection = rnd2 < params.sdTree.fraction ? newDirection1 : newDirection3;
        const auto  cosine = rnd2 < params.sdTree.fraction ? cosine1 : cosine3;
        const auto  bsdfPdf = getValPhongPDF(newDirection, reflectDir, mrec.shinness);
        const auto  bsdfVal = mrec.specular * bsdfPdf / a_specular;
        //両方とも正なら
        if (params.isBuilt)
        {
            const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
            const auto  woPdf = params.sdTree.fraction * dTreePdf + (1.0f - params.sdTree.fraction) * bsdfPdf;
            //printf("Hit1! %f %f\n", woPdf,dTreePdf);
            prd->dTreePdf = dTreePdf;
            prd->bsdfPdf = bsdfPdf;
            prd->woPdf = woPdf;
            prd->bsdfVal = bsdfVal;
            prd->throughPut *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine = cosine;
            setRayDirection(newDirection);
        }
        else {
            prd->bsdfPdf = getValPhongPDF(newDirection3, reflectDir, mrec.shinness);
            prd->dTreePdf = 0.0f;
            prd->woPdf = prd->bsdfPdf;
            prd->bsdfVal = (mrec.specular * prd->bsdfPdf / a_specular);
            prd->throughPut *= (mrec.specular * rtlib::max(cosine3, 0.0f) / a_specular);
            prd->cosine = cosine3;
            setRayDirection(newDirection3);
        }
        prd->countEmitted = true;
    }
    else {
        //printf("Hit!\n");
        //反射しない
        prd->bsdfVal = make_float3(1.0f);
        prd->woPdf = 0.0f;
        prd->bsdfPdf = 0.0f;
        prd->dTreePdf = 0.0f;
        prd->cosine = 0.0f;
        prd->throughPut = make_float3(0.0f);
        prd->dTree = nullptr;
        prd->done = true;
    }
    prd->seed = xor32.m_seed;
}
//specular
extern "C" __global__ void __closesthit__radiance_for_specular() {
    auto* hgData  = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec    = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec    = hgData->matParams.GetRecord(srec.texCoord);

    RadiancePRD* prd = getRadiancePRD();
    prd->dTree    = nullptr;
    prd->radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd->distance = optixGetRayTmax();
    {
        float3 reflectDir = rtlib::normalize(rtlib::reflect(rayDirection, srec.sNormal));
        auto cosine = rtlib::dot(reflectDir, srec.sNormal);

        prd->woPdf = 0.0f;
        prd->dTreePdf = 0.0f;
        prd->bsdfPdf = std::fabsf(cosine);

        setRayOrigin(srec.position);
        setRayDirection(reflectDir);
        prd->cosine = cosine;

        prd->bsdfVal = mrec.specular;
        prd->throughPut *= prd->bsdfVal;
        prd->countEmitted = true;
        prd->isDelta      = true;
    }
}
//reflection
extern "C" __global__ void __closesthit__radiance_for_refraction() {
    auto* hgData       = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec         = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec         = hgData->matParams.GetRecord(srec.texCoord);

    auto  refrInd      = mrec.refrInd;
    auto  normal       = srec.vNormal;
    if (rtlib::dot(normal, rayDirection) < 0.0f) {
        refrInd = 1.0f / refrInd;
    }
    else {
        normal = make_float3(-normal.x, -normal.y, -normal.z);
    }
    RadiancePRD* prd = getRadiancePRD();
    prd->dTree       = nullptr;
    prd->radiance    = make_float3(0.0f, 0.0f, 0.0f);
    prd->distance    = optixGetRayTmax();
    rtlib::Xorshift32 xor32(prd->seed);
    {
        float3 reflectDir = rtlib::normalize(rtlib::reflect(rayDirection, normal));
        float  cosine_i   =-rtlib::dot(normal, rayDirection);
        float  sine_o_2   = (1.0f - rtlib::pow2(cosine_i)) * rtlib::pow2(refrInd);
        float  f0         = rtlib::pow2((1 - refrInd) / (1 + refrInd));
        float  fresnell   = f0 + (1.0f - f0) * rtlib::pow5(1.0f - cosine_i);

        if (rtlib::random_float1(0.0f, 1.0f, xor32) < fresnell || sine_o_2 > 1.0f) {
            float cosine = rtlib::dot(reflectDir, normal);
            prd->woPdf   = prd->dTreePdf = 0.0f;
            prd->bsdfPdf = std::fabsf(cosine);
            setRayOrigin(srec.position + 0.001f * normal);
            setRayDirection(reflectDir);
            prd->cosine  = cosine;
            prd->bsdfVal = mrec.specular;
            prd->throughPut *= prd->bsdfVal;
        }
        else {
            float  cosine_o   = sqrtf(1.0f - sine_o_2);
            float3 k          = (rayDirection + cosine_i * normal) / sqrtf(1.0f - cosine_i * cosine_i);
            float3 refractDir = rtlib::normalize(sqrtf(sine_o_2) * k - cosine_o * normal);
            float cosine      = rtlib::dot(refractDir, normal);
            prd->woPdf        = prd->dTreePdf = 0.0f;
            prd->bsdfPdf      = std::fabsf(cosine);
            setRayOrigin(srec.position - 0.001f * normal);
            setRayDirection(refractDir);
            prd->cosine       = cosine;
            prd->bsdfVal      = make_float3(1.0f);
            prd->throughPut  *= prd->bsdfVal;
        }
        prd->isDelta = true;
    }
    prd->countEmitted = true;
    prd->seed = xor32.m_seed;
}
//emission
extern "C" __global__ void __closesthit__radiance_for_emission() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto  srec = getSurfaceRec(hgData->surfParams);
    auto  rayDirection = optixGetWorldRayDirection();
    auto  mrec = hgData->matParams.GetRecord(srec.texCoord);
    RadiancePRD* prd = getRadiancePRD();
    bool countEmitted = prd->countEmitted || ((mrec.flags & MATERIAL_FLAG_USE_NEE_BIT) == 0);
    prd->radiance = mrec.emission * prd->throughPut * static_cast<float>(countEmitted) * static_cast<float>(rtlib::dot(srec.sNormal, rayDirection) < 0.0f);
    prd->bsdfVal  = make_float3(1.0f);
    prd->woPdf    = 0.0f;
    prd->bsdfPdf  = 0.0f;
    prd->dTreePdf = 0.0f;
    prd->dTree    = nullptr;
    prd->cosine   = 0.0f;
    prd->distance = srec.distance;
    prd->done = true;
}
//occluded
extern "C" __global__ void __closesthit__occluded() {
    setPayloadOccluded(true);
}
