#include "hip/hip_runtime.h"
#define __HIPCC__
#include "RayTrace.h"
extern "C" {
    __constant__ Params params;
}
static __forceinline__ __device__ float3 faceForward(const float3& n, const float3& i, const float3& nref) {
    return copysignf(1.0f, rtlib::dot(n, i)) * nref;
}
static __forceinline__ __device__ void*  unpackPointer(unsigned int p0, unsigned int p1) {
    return reinterpret_cast<void*>(rtlib::to_combine(p0, p1));
}
static __forceinline__ __device__ void   packPointer(void* ptr,unsigned int& p0, unsigned int& p1) {
    const unsigned long long llv = reinterpret_cast<const unsigned long long>(ptr);
    p0 = rtlib::to_upper(llv);
    p1 = rtlib::to_lower(llv);
}
static __forceinline__ __device__ RadiancePRD* getRadiancePRD() {
    unsigned int p0 = optixGetPayload_0();
    unsigned int p1 = optixGetPayload_1();
    return static_cast<RadiancePRD*>(unpackPointer(p0, p1));
}
static __forceinline__ __device__ void setRadiancePRD(RadiancePRD* prd) {
    unsigned int p0;
    unsigned int p1;
    packPointer(static_cast<void*>(prd), p0, p1);
    optixSetPayload_0(p0);
    optixSetPayload_1(p1);
}
static __forceinline__ __device__ void  setPayloadOccluded(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}
static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    const float3& rayOrigin, 
    const float3& rayDirection,
    float tmin, float tmax,
    RadiancePRD*  prd) {
    unsigned int p0, p1;
    packPointer(prd, p0, p1);
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, RAY_TYPE_RADIANCE, RAY_TYPE_COUNT, RAY_TYPE_RADIANCE, p0, p1);
}
static __forceinline__ __device__ bool traceOccluded(
    OptixTraversableHandle handle,
    const float3& rayOrigin,
    const float3& rayDirection,
    float tmin, float tmax) {
    unsigned int occluded = false;
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, RAY_TYPE_OCCLUSION, RAY_TYPE_COUNT, RAY_TYPE_OCCLUSION, occluded);
    return occluded;
}
extern "C" __global__ void     __raygen__rg(){
    const uint3 idx             = optixGetLaunchIndex();
	const uint3 dim             = optixGetLaunchDimensions();
    auto* rgData                = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u              = rgData->u;
	const float3 v              = rgData->v;
	const float3 w              = rgData->w;
    unsigned int seed           = params.seed[params.width * idx.y + idx.x];
    float3 result               = make_float3(0.0f, 0.0f, 0.0f);
    size_t i                    = params.samplePerLaunch;
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d      = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed                = xor32.m_seed;
        float3 rayOrigin    = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.emitted         = make_float3(0.0f, 0.0f, 0.0f);
        prd.radiance        = make_float3(0.0f, 0.0f, 0.0f);
        prd.attenuation     = make_float3(1.0f, 1.0f, 1.0f);
        prd.countEmitted    = true;
        prd.done            = false;
        prd.seed            = seed;
        int depth = 0;
        for (;;) {
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            result += prd.emitted;
            result += prd.radiance * prd.attenuation;
            if (prd.done || depth >= TEST_MAX_TRACE_DEPTH) {
                break;
            }
            rayOrigin    = prd.origin;
            rayDirection = prd.direction;
            depth++;
        }
        seed = prd.seed;
    } while(--i);
    const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
    const float3 accumColor     = prevAccumColor + result;
    float3 frameColor           = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
    frameColor                  = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
    //if (idx.x == 500 && idx.y  == 500) {
        //printf("%f %f %f\n", frameColor.x, frameColor.y, frameColor.z);
    //}
    params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
        static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
        static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
        static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
    params.seed[params.width * idx.y + idx.x]        = seed;
}
extern "C" __global__ void       __miss__radiance(){
    auto* msData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    RadiancePRD* prd = getRadiancePRD();
    prd->radiance    = make_float3(msData->bgColor.x, msData->bgColor.y, msData->bgColor.z);
    prd->done        = true;
}
extern "C" __global__ void       __miss__occluded() {
    setPayloadOccluded(false);
}
extern "C" __global__ void __closesthit__radiance_for_diffuse()  {
    auto*        hgData       = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID  = optixGetPrimitiveIndex();
    const float3 v0           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    const float3 n0           = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal       = faceForward(n0, make_float3(-rayDirection.x,-rayDirection.y,-rayDirection.z), n0);
    const float2 barycentric  = optixGetTriangleBarycentrics();
    const auto t0             = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1             = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2             = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord       = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position     = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd          = getRadiancePRD();
    prd->emitted              = make_float3(0.0f, 0.0f, 0.0f);
    rtlib::Xorshift32 xor32(prd->seed);
    {
        rtlib::ONB onb(normal);
        float3 newDirection = onb.local(rtlib::random_cosine_direction(xor32));
        prd->direction      = newDirection;
        prd->origin         = position;
        float3 diffuse      = hgData->getDiffuseColor(texCoord);
        float3 specular     = hgData->getSpecularColor(texCoord);
        float3 reflectDir   = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);
        float  shinness     = hgData->shinness;
        float cosine        = fabsf(rtlib::dot(newDirection, reflectDir));
        prd->attenuation   *= diffuse + specular * (shinness + 2.0f) * powf(cosine, shinness) / 2.0f;
        prd->countEmitted   = false;
    }
    {
        const float2 z        = rtlib::random_float2(xor32);
        const auto   light    = params.light;
        const float3 lightPos = light.corner + light.v1 * z.x + light.v2 * z.y;
        const float  Ldist    = rtlib::distance(lightPos, position);
        const float3 lightDir = rtlib::normalize(lightPos - position);
        const float  ndl      = rtlib::dot(normal, lightDir);
        const float  lndl     =-rtlib::dot(light.normal, lightDir);
        float weight = 0.0f;
        if (ndl > 0.0f && lndl > 0.0f) {
            const bool occluded = traceOccluded(params.gasHandle, position, lightDir, 0.01f, Ldist - 0.01f);
            if (!occluded) {
                //printf("not Occluded!\n");
                const float A = rtlib::length(rtlib::cross(light.v1, light.v2));
                weight = ndl * lndl * A / (RTLIB_M_PI * Ldist * Ldist);
            }
        }
        prd->radiance += light.emission * weight;
        
    }
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_specular() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID  = optixGetPrimitiveIndex();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);
    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd = getRadiancePRD();
    prd->emitted     = make_float3(0.0f, 0.0f, 0.0f);
    {
        float3 specular   = hgData->getSpecularColor(texCoord);
        float3 reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);
        prd->origin       = position;
        prd->direction    = reflectDir;
        prd->attenuation *= specular;
        prd->countEmitted = true;
    }
}
extern "C" __global__ void __closesthit__radiance_for_refraction() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    float3 normal   = {};
    float  refInd   = 0.0f;
    if (rtlib::dot(n0,rayDirection)<0.0f) {
        normal      = n0;
        refInd      = 1.0f / hgData->refrInd; 
    }
    else {
        normal      = make_float3(-n0.x,-n0.y,-n0.z);
        refInd      = hgData->refrInd;
    }
    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd      = getRadiancePRD();
    prd->emitted          = make_float3(0.0f, 0.0f, 0.0f);
    rtlib::Xorshift32 xor32(prd->seed);
    float3 diffuse        = hgData->getDiffuseColor(texCoord);
    float3 specular       = hgData->getSpecularColor(texCoord);
    float3 transmit       = hgData->transmit;
    {
        prd->origin       = position;
        float3 reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);
        float  cosine_i   = -rtlib::dot(normal, rayDirection);
        float  sine_o_2   = (1.0f - rtlib::pow2(cosine_i)) * rtlib::pow2(refInd);
        float  f0         = rtlib::pow2((1 - refInd) / (1 + refInd));
        float  fresnell   = f0 + (1.0f - f0) * rtlib::pow5(1.0f - cosine_i);
        if (rtlib::random_float1(0.0f, 1.0f, xor32) < fresnell || sine_o_2 > 1.0f) {

            //printf("reflect: %lf %lf %lf\n", reflectDir.x, reflectDir.y, reflectDir.z);
            prd->origin      += 0.001f * normal;
            prd->direction    = reflectDir;
            prd->attenuation *= specular;
        }
        else {
            float  cosine_o   = sqrtf(1.0f - sine_o_2);
            float3 refractDir = (rayDirection - (cosine_o - cosine_i) * normal) / refInd;
            //printf("refract: %lf %lf %lf\n", refractDir.x, refractDir.y, refractDir.z);
            prd->origin      -= 0.001f * normal;
            prd->direction    = refractDir;
            prd->attenuation *= transmit;
        }
        prd->countEmitted   = true;
    }
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_emission() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    //const float3 n0 = rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0));
    //const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);
    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord   = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd      = getRadiancePRD();
    if (prd->countEmitted) {
        prd->emitted = hgData->getEmissionColor(texCoord) * prd->attenuation;
    }
    prd->countEmitted = false;
}
extern "C" __global__ void __closesthit__occluded() {
    setPayloadOccluded(true);
}
