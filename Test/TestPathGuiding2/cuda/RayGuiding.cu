#include "hip/hip_runtime.h"
#define __HIPCC__
#include "RayTrace.h"
#include "PathGuiding.h"
struct RadiancePRD {
    DTreeWrapper* dTree;
    float3        dTreeVoxelSize;
    float3        emission;
    float3        bsdfVal;
    float3        throughPut;
    float         woPdf, bsdfPdf, dTreePdf;
    float         cosine;
    float         distance;
    unsigned int  seed;
    bool          isDelta;
    bool          done;
};
extern "C" {
    __constant__ RayTraceParams params;
}
static __forceinline__ __device__ float3       faceForward(const float3& n, const float3& i, const float3& nref) {
    return copysignf(1.0f, rtlib::dot(n, i)) * nref;
}
static __forceinline__ __device__ void*        unpackPointer(unsigned int p0, unsigned int p1) {
    return reinterpret_cast<void*>(rtlib::to_combine(p0, p1));
}
static __forceinline__ __device__ void         packPointer(void* ptr, unsigned int& p0, unsigned int& p1) {
    const unsigned long long llv = reinterpret_cast<const unsigned long long>(ptr);
    p0 = rtlib::to_upper(llv);
    p1 = rtlib::to_lower(llv);
}
static __forceinline__ __device__ float3       unpackFloat3(unsigned int p0, unsigned p1, unsigned int p2)
{
    return make_float3(__uint_as_float(p0), __uint_as_float(p1), __uint_as_float(p2));
}
static __forceinline__ __device__ void         packFloat3(const float3& v, unsigned int& p0, unsigned& p1, unsigned int& p2)
{
    p0 = __float_as_uint(v.x);
    p1 = __float_as_uint(v.y);
    p2 = __float_as_uint(v.z);
}
static __forceinline__ __device__ RadiancePRD* getRadiancePRD() {
    unsigned int p0 = optixGetPayload_0();
    unsigned int p1 = optixGetPayload_1();
    return static_cast<RadiancePRD*>(unpackPointer(p0, p1));
}
static __forceinline__ __device__ void         setRadiancePRD(RadiancePRD* prd) {
    unsigned int p0;
    unsigned int p1;
    packPointer(static_cast<void*>(prd), p0, p1);
    optixSetPayload_0(p0);
    optixSetPayload_1(p1);
}
static __forceinline__ __device__ void         setRayOrigin(const float3& origin) {
    unsigned int p2, p3, p4;
    packFloat3(origin, p2, p3, p4);
    optixSetPayload_2(p2);
    optixSetPayload_3(p3);
    optixSetPayload_4(p4);
}
static __forceinline__ __device__ void         setRayDirection(const float3& direction) {
    unsigned int p5, p6, p7;
    packFloat3(direction, p5, p6, p7);
    optixSetPayload_5(p5);
    optixSetPayload_6(p6);
    optixSetPayload_7(p7);
}
static __forceinline__ __device__ void         setPayloadOccluded(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}
static __forceinline__ __device__ void         traceRadiance(
    OptixTraversableHandle handle,
    float3& rayOrigin,
    float3& rayDirection,
    float tmin, float tmax,
    RadiancePRD* prd) {
    unsigned int p0, p1, p2, p3, p4, p5, p6, p7;
    packPointer(prd, p0, p1);
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, RAY_TYPE_RADIANCE, RAY_TYPE_COUNT, RAY_TYPE_RADIANCE, p0, p1, p2, p3, p4, p5, p6, p7);
    rayOrigin = unpackFloat3(p2, p3, p4);
    rayDirection = unpackFloat3(p5, p6, p7);
}
static __forceinline__ __device__ bool         traceOccluded(
    OptixTraversableHandle handle,
    const float3& rayOrigin,
    const float3& rayDirection,
    float tmin, float tmax) {
    unsigned int occluded = false;
    optixTrace(handle, rayOrigin, rayDirection, tmin, tmax, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, RAY_TYPE_OCCLUSION, RAY_TYPE_COUNT, RAY_TYPE_OCCLUSION, occluded);
    return occluded;
}
extern "C" __global__ void __raygen__def() {

    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    auto* rgData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u = rgData->u;
    const float3 v = rgData->v;
    const float3 w = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.emission   = make_float3(0.0f);
        prd.bsdfVal    = make_float3(1.0f);
        prd.throughPut = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf      = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.cosine     = 0.0f;
        prd.distance   = 0.0f;
        prd.done       = false;
        prd.isDelta    = false;
        prd.seed       = seed;
        int depth      = 0;
        for (;;) {
            float3 prvThroughPut = prd.throughPut;
            //
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            //vertices�̍X�V
            //Radiance�̍X�V
            //Result�̍X�V
            result += prvThroughPut * prd.emission;
            //ThroughPut�̍X�V
            if (prd.done || depth >= params.maxTraceDepth) {
                break;
            }
            depth++;
        }
        seed = prd.seed;
    } while (i--);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor     = prevAccumColor + result;
        float3 frameColor           = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor                  = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
        params.frameBuffer[params.width * idx.y + idx.x] = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __raygen__pg() {

    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    auto* rgData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const float3 u = rgData->u;
    const float3 v = rgData->v;
    const float3 w = rgData->w;
    unsigned int seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    size_t i = params.samplePerLaunch;
    TraceVertex vertices[RAY_TRACE_MAX_VERTEX_COUNT] = {};
    do {
        rtlib::Xorshift32 xor32(seed);
        const float2 jitter = rtlib::random_float2(xor32);
        const float2 d = make_float2(
            ((2.0f * static_cast<float>(idx.x) + jitter.x) / static_cast<float>(dim.x)) - 1.0,
            ((2.0f * static_cast<float>(idx.y) + jitter.y) / static_cast<float>(dim.y)) - 1.0);
        seed = xor32.m_seed;
        float3 rayOrigin = rgData->eye;
        float3 rayDirection = rtlib::normalize(d.x * u + d.y * v + w);
        RadiancePRD prd;
        prd.emission   = make_float3(0.0f);
        prd.bsdfVal    = make_float3(1.0f);
        prd.throughPut = make_float3(1.0f);
        prd.dTreeVoxelSize = make_float3(1.0f);
        prd.woPdf = prd.bsdfPdf = prd.dTreePdf = 0.0f;
        prd.distance   = 0.0f;
        prd.done       = false;
        prd.isDelta    = false;
        prd.seed       = seed;
        int depth      = 0;
        for (;;) {
            float3 prvThroughPut           = prd.throughPut;
            traceRadiance(params.gasHandle, rayOrigin, rayDirection, 0.01f, 1e16f, &prd);
            vertices[depth].rayOrigin      = rayOrigin;
            vertices[depth].rayDirection   = rayDirection;
            vertices[depth].dTree          = prd.dTree;
            vertices[depth].dTreeVoxelSize = prd.dTreeVoxelSize;
            vertices[depth].throughPut     = prd.throughPut;
            vertices[depth].bsdfVal        = prd.bsdfVal;
            vertices[depth].radiance       = make_float3(0.0f);
            vertices[depth].woPdf          = prd.woPdf;
            vertices[depth].bsdfPdf        = prd.bsdfPdf;
            vertices[depth].dTreePdf       = prd.dTreePdf;
            vertices[depth].cosine         = prd.cosine;
            vertices[depth].isDelta        = prd.isDelta;
            for (int j = 0; j < depth; ++j) {
                vertices[j].Record(prvThroughPut * prd.emission );
            }
            //OK
            //Result�̍X�V
            result += prvThroughPut * prd.emission;
            //if (isnan(result.x) || isnan(result.y) || isnan(result.z)) {
              //  printf("result is nan Bug(%d,%d,%d): Result(%f %f %f) prvThroughPut(%lf %lf %lf) prd.emission(%lf %lf %lf) prd.pdf: (%lf %lf %lf) prd.done: %d\n",
              //      idx.x, idx.y, depth,
              //      result.x, result.y, result.z,
              //      prvThroughPut.x, prvThroughPut.y, prvThroughPut.z,
              //      prd.emission.x , prd.emission.y , prd.emission.z ,
              //      prd.woPdf, prd.bsdfPdf, prd.dTreePdf, (int)prd.done
              //  );
            //}
            bool isValidThroughPut =  ( !isnan(prd.throughPut.x) &&   !isnan(prd.throughPut.y) &&   !isnan(prd.throughPut.z)  &&
                                      isfinite(prd.throughPut.x) && isfinite(prd.throughPut.y) && isfinite(prd.throughPut.z)  &&
                                      (prd.throughPut.x >= 0.0f  && prd.throughPut.y >= 0.0f   && prd.throughPut.z >= 0.0f));
            //ThroughPut�̍X�V
            //if (!isValidThroughPut) {
              // printf("prd.ThroughPut is Invalid Bug(%d,%d,%d): Result(%f %f %f)  prd.throughPut(%lf %lf %lf) prd.emission(%lf %lf %lf) prd.pdf: (%lf %lf %lf) prd.done: %d\n",
              //      idx.x, idx.y, depth,
              //     result.x, result.y, result.z,
              //     prd.throughPut.x, prd.throughPut.y, prd.throughPut.z,
              //     prd.emission.x, prd.emission.y, prd.emission.z,
              //     prd.woPdf, prd.bsdfPdf, prd.dTreePdf, (int)prd.done
              // );
           //}
            if (prd.done || depth >= params.maxTraceDepth || !isValidThroughPut) {
                break;
            }
            depth++;
        }
        for (int j = 0; j < depth; ++j) {
            vertices[j].Commit<RAY_TRACE_S_FILTER, RAY_TRACE_D_FILTER>(params.sdTree,1.0f);
        }
        seed = prd.seed;
    } while (i--);
    {
        const float3 prevAccumColor = params.accumBuffer[params.width * idx.y + idx.x];
        const float3 accumColor = prevAccumColor + result;
        float3 frameColor = accumColor / (static_cast<float>(params.samplePerALL + params.samplePerLaunch));
        frameColor = frameColor / (make_float3(1.0f, 1.0f, 1.0f) + frameColor);
        params.accumBuffer[params.width * idx.y + idx.x] = accumColor;
    }
    {
        const float3 prevAccumColor2 = params.accumBuffer2[params.width * idx.y + idx.x];
        const float3 accumColor2 = prevAccumColor2 + result;
        float3       frameColor2 = accumColor2 / (static_cast<float>(params.samplePerALL2 + params.samplePerLaunch));
        frameColor2 = frameColor2 / (make_float3(1.0f, 1.0f, 1.0f) + frameColor2);
        params.accumBuffer2[params.width * idx.y + idx.x] = accumColor2;
        params.frameBuffer[params.width * idx.y + idx.x]  = make_uchar4(
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor2.x)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor2.y)),
            static_cast<unsigned char>(255.99 * rtlib::linear_to_gamma(frameColor2.z)), 255);
    }
    params.seedBuffer[params.width * idx.y + idx.x] = seed;
}
extern "C" __global__ void __miss__radiance() {
    auto* msData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    RadiancePRD* prd = getRadiancePRD();

    prd->emission = make_float3(msData->bgColor.x, msData->bgColor.y, msData->bgColor.z);
    prd->dTree    = nullptr;
    prd->woPdf    = 1.0f;
    prd->bsdfPdf  = 0.0f;
    prd->dTreePdf = 0.0f;
    prd->bsdfVal  = make_float3(1.0f);
    prd->cosine   = 0.0f;
    prd->distance = optixGetRayTmax();
    prd->done     = true;
}
extern "C" __global__ void __miss__occluded() {
    setPayloadOccluded(false);
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID  = optixGetPrimitiveIndex();

    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0          = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal      = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0            = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1            = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2            = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord      = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;

    const auto diffuse       = hgData->getDiffuseColor(texCoord);
    const auto emission      = hgData->getEmissionColor(texCoord);

    const auto distance      = optixGetRayTmax();
    const float3 position    = optixGetWorldRayOrigin() + distance * rayDirection;
    float3 newDirection      = make_float3(0.0f);
    RadiancePRD* prd         = getRadiancePRD();

    prd->dTree    = nullptr;
    prd->emission = emission;
    prd->distance = distance;

    rtlib::Xorshift32 xor32(prd->seed);
    rtlib::ONB onb(normal);
    newDirection = onb.local(rtlib::random_cosine_direction(xor32));

    const auto cosine = rtlib::dot(newDirection, normal);

    prd->bsdfPdf  = fabsf(cosine) / RTLIB_M_PI;
    prd->dTreePdf = 0.0f;
    prd->woPdf    = prd->bsdfPdf;

    setRayOrigin(position);
    setRayDirection(newDirection);

    prd->cosine      = cosine;
    prd->bsdfVal     = diffuse / RTLIB_M_PI;
    prd->throughPut *= diffuse;
    prd->seed        = xor32.m_seed;
    prd->isDelta     = false;
}
extern "C" __global__ void __closesthit__radiance_for_diffuse_pg() {

    auto* hgData              = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID  = optixGetPrimitiveIndex();

    const float3 v0           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0           = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal       = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric  = optixGetTriangleBarycentrics();
    const auto t0             = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1             = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2             = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord       = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;

    const auto diffuse   = hgData->getDiffuseColor( texCoord);
    const auto emission  = hgData->getEmissionColor(texCoord);

    const auto distance  = optixGetRayTmax();
    const auto position  = optixGetWorldRayOrigin() + distance * rayDirection;
    auto dTreeVoxelSize  = make_float3(0.0f);
    const auto dTree     = params.sdTree.GetDTreeWrapper(position, dTreeVoxelSize);

    float3 newDirection1 = make_float3(0.0f);
    float3 newDirection2 = make_float3(0.0f);
    float  cosine1       = 0.0f;
    float  cosine2       = 0.0f;

    RadiancePRD* prd    = getRadiancePRD();
    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->emission       = emission;
    prd->distance       = distance;
    prd->bsdfPdf        = 0.0f;
    prd->bsdfVal        = diffuse / RTLIB_M_PI;
    prd->isDelta        = false;
    rtlib::Xorshift32 xor32(prd->seed);

    setRayOrigin(position);
    {
#if defined(RAY_GUIDING_SAMPLE_BY_UNIFORM_SPHERE)
        rtlib::ONB onb(normal);
        newDirection1 = onb.local(rtlib::random_in_unit_sphere(xor32));
        cosine1 = rtlib::dot(normal, newDirection1);
#elif defined(RAY_GUIDING_SAMPLE_BY_COSINE_SPHERE)
        rtlib::ONB onb(normal);
        newDirection1 = onb.local(rtlib::random_cosine_direction(xor32));
        cosine1       = rtlib::dot(normal, newDirection1);
#else
        newDirection1 = dTree->Sample(xor32);
        cosine1       = rtlib::dot(normal, newDirection1);
#endif

        if (isnan(newDirection1.x) || isnan(newDirection1.y) || isnan(newDirection1.z)) {
            printf("newDirection1 is nan: new Direction1 = (%f, %f, %f) normal = (%f, %f, %f) n0 = (%f, %f, %f)\n", newDirection1.x, newDirection1.y, newDirection1.z, normal.x, normal.y, normal.z, n0.x, n0.y, n0.z);
        }
    }
    {
        rtlib::ONB onb(normal);
        newDirection2 = onb.local(rtlib::random_cosine_direction(xor32));
        cosine2       = rtlib::dot(normal, newDirection2);
        if (isnan(newDirection2.x) || isnan(newDirection2.y) || isnan(newDirection2.z))
        {
            printf("newDirection2 is nan!\n");
        }
    }


    const float rnd          = rtlib::random_float1(xor32);
    const auto  newDirection = rnd < 0.5f ? newDirection1 : newDirection2;
    const auto  cosine       = rnd < 0.5f ? cosine1 : cosine2;
    const auto  bsdfPdf      = rtlib::max(cosine / RTLIB_M_PI     ,0.0f);
    const auto  dTreePdf     = rtlib::max(dTree->Pdf(newDirection),0.0f);
    const auto  woPdf        = 0.5f * bsdfPdf + 0.5f * dTreePdf;
    //両方とも正なら
    if (params.isBuilt && woPdf > 0.0f) {
        prd->bsdfPdf     = bsdfPdf;
        prd->dTreePdf    = dTreePdf;
        prd->woPdf       = woPdf;
        prd->throughPut *= (prd->bsdfVal * rtlib::max(cosine,0.0f) / woPdf);
        setRayDirection(newDirection);
        prd->cosine      = cosine;
    }
    else {
        prd->bsdfPdf     = fabsf(cosine2) / RTLIB_M_PI;
        prd->dTreePdf    = 0.0f;
        prd->woPdf       = prd->bsdfPdf;
        prd->throughPut *= (diffuse);
        setRayDirection(newDirection2);
        prd->cosine      = cosine2;
    }
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_specular() {

    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);
    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd = getRadiancePRD();
    prd->dTree = nullptr;
    prd->emission = make_float3(0.0f, 0.0f, 0.0f);
    prd->distance = optixGetRayTmax();
    {
        float3 specular   = hgData->getSpecularColor(texCoord);
        float3 reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);
        auto cosine       = rtlib::dot(reflectDir, normal);

        prd->woPdf        = 0.0f;
        prd->dTreePdf     = 0.0f;
        prd->bsdfPdf      = std::fabsf(cosine);

        setRayOrigin(position);
        setRayDirection(reflectDir);
        prd->cosine      = cosine;

        prd->bsdfVal     = specular;
        prd->throughPut *= prd->bsdfVal;
        prd->isDelta     = true;
    }
}
extern "C" __global__ void __closesthit__radiance_for_refraction() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    float3 normal = {};
    float  refInd = 0.0f;
    if (rtlib::dot(n0, rayDirection) < 0.0f) {
        normal = n0;
        refInd = 1.0f / hgData->refrInd;
    }
    else {
        normal = make_float3(-n0.x, -n0.y, -n0.z);
        refInd = hgData->refrInd;
    }
    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const float3 position = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDirection;
    RadiancePRD* prd = getRadiancePRD();
    prd->dTree    = nullptr;
    prd->emission = make_float3(0.0f, 0.0f, 0.0f);
    prd->distance = optixGetRayTmax();
    rtlib::Xorshift32 xor32(prd->seed);
    float3 diffuse  = hgData->getDiffuseColor(texCoord);
    float3 specular = hgData->getSpecularColor(texCoord);
    float3 transmit = hgData->transmit;
    {
        float3 reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);
        float  cosine_i = -rtlib::dot(normal, rayDirection);
        float  sine_o_2 = (1.0f - rtlib::pow2(cosine_i)) * rtlib::pow2(refInd);
        float  f0 = rtlib::pow2((1 - refInd) / (1 + refInd));
        float  fresnell = f0 + (1.0f - f0) * rtlib::pow5(1.0f - cosine_i);

        if (rtlib::random_float1(0.0f, 1.0f, xor32) < fresnell || sine_o_2 > 1.0f) {
            float cosine     = rtlib::dot(reflectDir, normal);
            prd->woPdf       = prd->dTreePdf = 0.0f;
            prd->bsdfPdf     = std::fabsf(cosine);
            //printf("reflect: %lf %lf %lf\n", reflectDir.x, reflectDir.y, reflectDir.z);
            setRayOrigin(position + 0.001f * normal);
            setRayDirection(reflectDir);
            prd->cosine      = cosine;
            prd->bsdfVal     = specular;
            prd->throughPut *= prd->bsdfVal;
        }
        else {
            float  cosine_o   = sqrtf(1.0f - sine_o_2);
            float3 refractDir = rtlib::normalize((rayDirection - (cosine_o - cosine_i) * normal) / refInd);
            float cosine = rtlib::dot(refractDir, normal);
            prd->woPdf   = prd->dTreePdf = 0.0f;
            prd->bsdfPdf = std::fabsf(cosine);
            //printf("refract: %lf %lf %lf\n", refractDir.x, refractDir.y, refractDir.z);
            setRayOrigin(position - 0.001f * normal);
            setRayDirection(refractDir);
            prd->cosine      = cosine;
            prd->bsdfVal     = make_float3(1.0f);
            prd->throughPut *= prd->bsdfVal;
        }
        prd->isDelta = true;
    }
    prd->seed = xor32.m_seed;
}
extern "C" __global__ void __closesthit__radiance_for_emission() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();
    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);
    //const float3 n0 = rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0));
    //const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);
    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];
    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const auto distance = optixGetRayTmax();
    const float3 position = optixGetWorldRayOrigin() + distance * rayDirection;
    RadiancePRD* prd = getRadiancePRD();
    prd->emission    = hgData->getEmissionColor(texCoord);
    prd->bsdfVal     = make_float3(1.0f);
    prd->woPdf       = 0.0f;
    prd->bsdfPdf     = 0.0f;
    prd->dTreePdf    = 0.0f;
    prd->dTree       = nullptr;
    prd->cosine      = 0.0f;
    prd->distance    = distance;
    prd->done        = true;
}
extern "C" __global__ void __closesthit__occluded() {
    setPayloadOccluded(true);
}
extern "C" __global__ void __closesthit__radiance_for_phong_def() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const float3 rayDirection = optixGetWorldRayDirection();
    const int    primitiveID = optixGetPrimitiveIndex();

    const float3 v0 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const float3 v1 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const float3 v2 = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const float3 n0 = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const float3 normal = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);

    const float2 barycentric = optixGetTriangleBarycentrics();
    const auto t0 = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1 = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2 = hgData->texCoords[hgData->indices[primitiveID].z];

    const auto reflectDir = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);

    const auto texCoord = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const auto diffuse  = hgData->getDiffuseColor(texCoord);
    const auto specular = hgData->getSpecularColor(texCoord);
    const auto shinness = hgData->shinness;
    const auto emission = hgData->getEmissionColor(texCoord);
    const auto distance = optixGetRayTmax();
    const float3 position = optixGetWorldRayOrigin() + distance * rayDirection;
    RadiancePRD* prd = getRadiancePRD();

    prd->dTree    = nullptr;
    prd->emission = emission;
    prd->distance = distance;

    rtlib::Xorshift32 xor32(prd->seed);

    auto  newDirection = make_float3(0.0f);
    auto  cosine = 0.0f;
    auto  weight = make_float3(0.0f);

    const auto rnd = rtlib::random_float1(xor32);
    const auto a_diffuse = (diffuse.x + diffuse.y + diffuse.z) / 3.0f;
    const auto a_specular = (specular.x + specular.y + specular.z) / 3.0f;
    if (rnd < a_diffuse) {
        rtlib::ONB onb(normal);
        newDirection  = onb.local(rtlib::random_cosine_direction(xor32));
        cosine        = rtlib::dot(newDirection, normal);
        prd->bsdfVal  = diffuse / (a_diffuse * RTLIB_M_PI);
        prd->bsdfPdf  = fabsf(cosine) / RTLIB_M_PI;
        prd->dTreePdf = 0.0f;
        prd->woPdf    = prd->bsdfPdf;
        prd->throughPut *= (diffuse/a_diffuse);
        prd->cosine   = cosine;
    }
    else if (rnd < a_diffuse + a_specular) {
        const auto cosTht = powf(rtlib::random_float1(0.0f, 1.0f, xor32), 1.0f / (shinness + 1.0f));
        const auto sinTht = sqrtf(1.0f - cosTht * cosTht);
        const auto phi = rtlib::random_float1(0.0f, RTLIB_M_2PI, xor32);
        rtlib::ONB onb(reflectDir);
        newDirection = onb.local(make_float3(sinTht * cosf(phi), sinTht * sinf(phi), cosTht));
        cosine       = rtlib::dot(newDirection, normal);
        prd->bsdfVal = (specular / a_specular) * (shinness + 2.0f) * powf(fabsf(rtlib::dot(reflectDir, newDirection)), shinness) / RTLIB_M_2PI;
        prd->bsdfPdf = rtlib::max((shinness + 2.0f) * powf(fabsf(rtlib::dot(reflectDir, newDirection)), shinness) / RTLIB_M_2PI, 1e-7f);
        prd->dTreePdf= 0.0f;
        prd->woPdf   = prd->bsdfPdf;
        prd->throughPut *= (specular * rtlib::max(cosine,0.0f) / a_specular);
        prd->cosine  = cosine;
    }

    setRayOrigin(position);
    setRayDirection(newDirection);
    prd->seed = xor32.m_seed;
    prd->isDelta = false;
}
extern "C" __global__ void __closesthit__radiance_for_phong_pg () {
    auto* hgData            = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const auto rayDirection = optixGetWorldRayDirection();
    const auto primitiveID  = optixGetPrimitiveIndex();

    const auto v0           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].x]);
    const auto v1           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].y]);
    const auto v2           = optixTransformPointFromObjectToWorldSpace(hgData->vertices[hgData->indices[primitiveID].z]);

    const auto n0           = optixTransformNormalFromObjectToWorldSpace(rtlib::normalize(rtlib::cross(v1 - v0, v2 - v0)));
    const auto normal       = faceForward(n0, make_float3(-rayDirection.x, -rayDirection.y, -rayDirection.z), n0);
    //if (isnan(normal.x) || isnan(normal.y) || isnan(normal.z)) {
    //    printf("normal is nan!\n");
    //}
     
    const auto barycentric  = optixGetTriangleBarycentrics();
    const auto t0           = hgData->texCoords[hgData->indices[primitiveID].x];
    const auto t1           = hgData->texCoords[hgData->indices[primitiveID].y];
    const auto t2           = hgData->texCoords[hgData->indices[primitiveID].z];


    const auto reflectDir   = rtlib::normalize(rayDirection - 2.0f * rtlib::dot(rayDirection, normal) * normal);

    const auto texCoord     = (1.0f - barycentric.x - barycentric.y) * t0 + barycentric.x * t1 + barycentric.y * t2;
    const auto emission     = hgData->getEmissionColor(texCoord);
    const auto diffuse      = hgData->getDiffuseColor( texCoord);
    const auto specular     = hgData->getSpecularColor(texCoord);
    const auto shinness     = hgData->shinness;
    const auto distance     = optixGetRayTmax();
    const auto position     = optixGetWorldRayOrigin() + distance * rayDirection;
    //direction
    float3 newDirection1    = make_float3(0.0f);
    float3 newDirection2    = make_float3(0.0f);
    float3 newDirection3    = make_float3(0.0f);
    //cosine
    float  cosine1      = 0.0f;
    float  cosine2      = 0.0f;
    float  cosine3      = 0.0f;
    //payLoad
    RadiancePRD* prd    = getRadiancePRD();
    auto dTreeVoxelSize = make_float3(0.0f);
    const auto dTree    = params.sdTree.GetDTreeWrapper(position, dTreeVoxelSize);
    prd->dTree          = dTree;
    prd->dTreeVoxelSize = dTreeVoxelSize;
    prd->emission       = emission;
    prd->distance       = distance;
    prd->isDelta        = false;
    rtlib::Xorshift32 xor32(prd->seed);
    //const auto isValid = false;
    setRayOrigin(position);
    {
#if defined(RAY_GUIDING_SAMPLE_BY_UNIFORM_SPHERE)
        rtlib::ONB onb(normal);
        newDirection1   = onb.local(rtlib::random_in_unit_sphere(xor32));
        cosine1         = rtlib::dot(normal, newDirection1);
#elif defined(RAY_GUIDING_SAMPLE_BY_COSINE_SPHERE)
        newDirection1   = rtlib::random_cosine_direction(xor32);
        cosine1         = rtlib::dot(normal, newDirection1);
#else
        newDirection1   = dTree->Sample(xor32);
        cosine1         = rtlib::dot(normal, newDirection1);
#endif
        //if (isnan(newDirection1.x) || isnan(newDirection1.y) || isnan(newDirection1.z))
        //{
        // printf("newDirection1 is nan!\n");
        //}
    }
    {
        rtlib::ONB onb(normal);
        newDirection2 = onb.local(rtlib::random_cosine_direction(xor32));
        cosine2       = rtlib::dot(normal, newDirection2);
        //if (isnan(newDirection2.x) || isnan(newDirection2.y) || isnan(newDirection2.z)) {
        //    printf("newDirection2 is nan: newDirection2 = (%f, %f, %f) normal = (%f, %f, %f) n0 = (%f, %f, %f)\n", newDirection2.x, newDirection2.y, newDirection2.z, normal.x, normal.y, normal.z, n0.x, n0.y, n0.z);
        //}
    }
    {
        rtlib::ONB onb(reflectDir);
        const auto cosTht = powf(rtlib::random_float1(0.0f, 1.0f, xor32), 1.0f / (shinness + 1.0f));
        const auto sinTht = sqrtf(1.0f - cosTht * cosTht);
        const auto phi    = rtlib::random_float1(0.0f, RTLIB_M_2PI, xor32);
        newDirection3     = onb.local(make_float3(sinTht * cosf(phi), sinTht * sinf(phi), cosTht));
        cosine3 = rtlib::dot(normal, newDirection3);

        //if (isnan(newDirection3.x) || isnan(newDirection3.y) || isnan(newDirection3.z))
        //{
        //    printf("newDirection3 is nan: newDirection3 = (%f, %f, %f) normal = (%f, %f, %f) n0 = (%f, %f, %f)\n", newDirection3.x, newDirection3.y, newDirection3.z, normal.x, normal.y, normal.z, n0.x, n0.y, n0.z);
        //}
    }
    const auto  a_diffuse    = ( diffuse.x +  diffuse.y +  diffuse.z) / 3.0f;
    const auto  a_specular   = (specular.x + specular.y + specular.z) / 3.0f;
    const float rnd1         = rtlib::random_float1(xor32);
    const float rnd2         = rtlib::random_float1(xor32);

    if (rnd1 < a_diffuse){
        const auto  newDirection = rnd2 < 0.5f ? newDirection1 : newDirection2;
        const auto  cosine       = rnd2 < 0.5f ?       cosine1 :       cosine2;
        const auto  bsdfPdf      = rtlib::max(            cosine/RTLIB_M_PI,0.0f);
#if defined(RAY_GUIDING_SAMPLE_BY_UNIFORM_SPHERE)
        const auto  dTreePdf = 1.0f / (4.0f * RTLIB_M_PI);
#elif defined(RAY_GUIDING_SAMPLE_BY_COSINE_SPHERE)
        const auto  dTreePdf = rtlib::max(newDirection.z / RTLIB_M_PI, 0.0f);
#else
        const auto  dTreePdf = rtlib::max(dTree->Pdf(newDirection), 0.0f);
#endif
        const auto  woPdf        = 0.5f * dTreePdf + 0.5f * bsdfPdf;
        const auto  bsdfVal      = diffuse / (RTLIB_M_PI * a_diffuse);
        //const auto  dTreePdf     = 1.0f / (4.0f * RTLIB_M_PI);
        //両方とも正なら
        if (params.isBuilt && woPdf > 0.0f) {
        //if (false){
            //printf("Hit1! %f %f\n", woPdf,dTreePdf);
            prd->bsdfVal       = bsdfVal;
            prd->dTreePdf      = dTreePdf;
            prd->bsdfPdf       = bsdfPdf;
            prd->woPdf         = woPdf;
            prd->throughPut   *= (bsdfVal * rtlib::max(cosine, 0.0f) / woPdf);
            prd->cosine        = cosine;
            setRayDirection(newDirection);
            //if (isnan(prd->throughPut.x) || isnan(prd->throughPut.y) || isnan(prd->throughPut.z)) {
              // printf("prd->weight0 is nan: %f %f %f\n", prd->woPdf, prd->bsdfPdf, prd->dTreePdf);
            //}
        }
        else {
            prd->bsdfVal       = (diffuse / (RTLIB_M_PI * a_diffuse));
            prd->bsdfPdf       = fabsf(cosine2) / RTLIB_M_PI;
            prd->dTreePdf      = 0.0f;
            prd->woPdf         = prd->bsdfPdf;
            prd->throughPut   *= (diffuse/ a_diffuse);
            prd->cosine        = cosine2;
            setRayDirection(newDirection2);
            //if (isnan(prd->throughPut.x) || isnan(prd->throughPut.y) || isnan(prd->throughPut.z)) {
                //printf("prd->weight1 is nan: %f %f %f\n", prd->woPdf, prd->bsdfPdf, prd->dTreePdf);
            //}
        }
    }
    else if (rnd1 < a_diffuse + a_specular)
    {
        const auto  newDirection = rnd2 < 0.5f ? newDirection1 : newDirection3;
        const auto  cosine       = rnd2 < 0.5f ?       cosine1 :       cosine3;
        const auto  bsdfPdf      = rtlib::max((shinness + 2.0f) * powf(rtlib::max(rtlib::dot(reflectDir, newDirection),0.0f), shinness)/ RTLIB_M_2PI,0.0f);
#if defined(RAY_GUIDING_SAMPLE_BY_UNIFORM_SPHERE)
        const auto  dTreePdf     = 1.0f / (4.0f * RTLIB_M_PI);
#elif defined(RAY_GUIDING_SAMPLE_BY_COSINE_SPHERE)
        const auto  dTreePdf     = rtlib::max(newDirection.z / RTLIB_M_PI,0.0f);
#else
        const auto  dTreePdf     = rtlib::max(dTree->Pdf(newDirection), 0.0f);
#endif
        const auto  woPdf        = 0.5f * dTreePdf + 0.5f * bsdfPdf;
        const auto  bsdfVal      = specular * bsdfPdf / a_specular;
        //両方とも正なら
        if (params.isBuilt && woPdf > 0.0f)
        //if(false)
        {
            //printf("Hit1! %f %f\n", woPdf,dTreePdf);
            prd->dTreePdf      = dTreePdf;
            prd->bsdfPdf       = bsdfPdf;
            prd->woPdf         = woPdf;
            prd->bsdfVal       = bsdfVal;
            prd->throughPut   *= (bsdfVal * rtlib::max(cosine,0.0f)/ woPdf);
            prd->cosine        = cosine;
            setRayDirection(newDirection);
            //if (isnan(prd->throughPut.x) || isnan(prd->throughPut.y) || isnan(prd->throughPut.z)) {
                //printf("prd->weight2 is nan: %f %f %f\n", prd->woPdf, prd->bsdfPdf, prd->dTreePdf);
            //}
        }
        else {
            //printf("Hit2!\n");
            const auto reflCos = rtlib::max(rtlib::dot(reflectDir, newDirection3), 0.0f);
            prd->bsdfPdf       = rtlib::max((shinness + 2.0f) * powf(reflCos, shinness) / RTLIB_M_2PI, 0.0f);
            prd->dTreePdf      = 0.0f;
            prd->woPdf         = prd->bsdfPdf;
            prd->bsdfVal       = (specular * prd->bsdfPdf   / a_specular);
            prd->throughPut   *= (specular * rtlib::max(cosine3, 0.0f) / a_specular);
            prd->cosine        = cosine3;
            setRayDirection(newDirection3);
            //if (isnan(prd->throughPut.x) || isnan(prd->throughPut.y) || isnan(prd->throughPut.z)) {
            //   printf("prd->weight3 is nan: (%f %f %f) reflCos = %f\n", prd->woPdf, prd->bsdfPdf, prd->dTreePdf, reflCos);
            //}
        }
    }
    else {
        //printf("Hit!\n");
        //反射しない
        prd->bsdfVal    = make_float3(1.0f);
        prd->woPdf      = 0.0f;
        prd->bsdfPdf    = 0.0f;
        prd->dTreePdf   = 0.0f;
        prd->cosine     = 0.0f;
        //prd->throughPut = make_float3(0.0f);
        prd->dTree      = nullptr;
        prd->done       = true;
    }
    
    prd->seed = xor32.m_seed;
}
